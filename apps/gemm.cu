#include <hip/hip_runtime.h>
#include <>
#include <gflags/gflags.h>

#include <fstream>
#include <iostream>
#include <list>
#include <utility>

#include "core/common/types.h"
#include "core/common/yaml_config.h"
#include "core/components/scheduler/scheduler.h"
#include "core/matrixgraph.cuh"
#include "core/task/gpu_task/gemm.cuh"
#include "core/task/gpu_task/task_base.cuh"

DEFINE_string(i, "", "input data dir path for graph.");
DEFINE_string(it, "", "input data dir path for transposed graph.");
DEFINE_string(o, "", "output path.");
DEFINE_int32(count, 1, "count");
DEFINE_string(scheduler, "CHBL", "scheduler type.");

using sics::matrixgraph::core::components::scheduler::SchedulerType;
using sics::matrixgraph::core::task::GEMM;

SchedulerType Scheduler2Enum(const std::string& s) {
  if (s == "EvenSplit")
    return sics::matrixgraph::core::components::scheduler::kEvenSplit;
  else if (s == "CHBL")
    return sics::matrixgraph::core::components::scheduler::kCHBL;
  else if (s == "RoundRobin")
    return sics::matrixgraph::core::components::scheduler::kRoundRobin;
  return sics::matrixgraph::core::components::scheduler::kCHBL;
};

int main(int argc, char* argv[]) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  auto scheduler_type = Scheduler2Enum(FLAGS_scheduler);
  sics::matrixgraph::core::MatrixGraph system(scheduler_type);

  auto* gemm = new GEMM(FLAGS_i, FLAGS_it, FLAGS_o, FLAGS_count);

  // State which application is going to be running.
  system.Run(sics::matrixgraph::core::common::kGEMM, gemm);

  gflags::ShutDownCommandLineFlags();
  return EXIT_SUCCESS;
}
