#include "hip/hip_runtime.h"
#include "core/data_structures/grid_tiled_matrix.cuh"

#include <algorithm>
#include <cmath>

#ifdef TBB_FOUND
#include <execution>
#endif

namespace sics {
namespace matrixgraph {
namespace core {
namespace data_structures {

using GraphID = sics::matrixgraph::core::common::GraphID;
using VertexID = sics::matrixgraph::core::common::VertexID;

GridTiledMatrix::GridTiledMatrix(const GridGraphMetadata &metadata) {
  metadata_ = metadata;
  tiled_matrix_vec_.resize(pow(metadata_.n_chunks, 2));

  std::generate(std::execution::par, tiled_matrix_vec_.begin(),
                tiled_matrix_vec_.end(), []() { return new BitTiledMatrix(); });
}

void GridTiledMatrix::Print() const {
  std::cout << "[GridTiledMatrix Print] " << metadata_.n_chunks << "x"
            << metadata_.n_chunks << std::endl;
  for (GraphID x = 0; x < metadata_.n_chunks; x++) {
    for (GraphID y = 0; y < metadata_.n_chunks; y++) {
      if (tiled_matrix_vec_[x * metadata_.n_chunks + y]
              ->GetMetadata()
              .n_nz_tile == 0) {
        continue;
      }
      std::cout << "Block (" << x << "," << y << ")" << std::endl;
      tiled_matrix_vec_[x * metadata_.n_chunks + y]->Print();
    }
  }
}

} // namespace data_structures
} // namespace core
} // namespace matrixgraph
} // namespace sics