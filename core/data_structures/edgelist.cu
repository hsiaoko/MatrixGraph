#include "hip/hip_runtime.h"
#include "core/data_structures/edgelist.h"

#include <execution>
#include <filesystem>
#include <fstream>
#include <numeric>
#include <thread>

#include "core/common/consts.h"
#include "core/common/types.h"
#include "core/util/atomic.h"
#include "core/util/bitmap.h"

namespace sics {
namespace matrixgraph {
namespace core {
namespace data_structures {

using VertexID = sics::matrixgraph::core::common::VertexID;
using EdgeIndex = sics::matrixgraph::core::common::EdgeIndex;
using sics::matrixgraph::core::util::atomic::WriteAdd;
using sics::matrixgraph::core::util::atomic::WriteMax;
using sics::matrixgraph::core::util::atomic::WriteMin;
using std::filesystem::create_directory;
using std::filesystem::exists;

Edges::Edges(const Edges &edges) {
  edgelist_metadata_ = edges.get_metadata();
  edges_ptr_ = new Edge[edgelist_metadata_.num_edges]();
  memcpy(edges_ptr_, edges.get_base_ptr(),
         sizeof(Edge) * edgelist_metadata_.num_edges);
  if (edges.get_localid_to_globalid_ptr() != nullptr) {
    memcpy(localid_to_globalid_, edges.get_localid_to_globalid_ptr(),
           sizeof(Edge) * edgelist_metadata_.num_vertices);
  }
}

Edges::Edges(EdgeIndex n_edges, VertexID *edges_buf,
             VertexID *localid2globalid) {
  Init(n_edges, edges_buf, localid2globalid);
}

void Edges::Init(EdgeIndex n_edges, VertexID *edges_buf,
                 VertexID *localid2globalid) {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::mutex mtx;
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  Bitmap bm(n_edges);
  edges_ptr_ = new Edge[n_edges]();
  VertexID max_vid = 0;
  VertexID min_vid = MAX_VERTEX_ID;

  // Get Min Max vertex ID.
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, n_edges, step, &edges_buf, &min_vid, &max_vid](auto w) {
                  for (EdgeIndex _ = w; _ < n_edges; _ += step) {
                    edges_ptr_[_].src = edges_buf[_ * 2];
                    edges_ptr_[_].dst = edges_buf[_ * 2 + 1];
                    WriteMin(&min_vid, edges_ptr_[_].src);
                    WriteMin(&min_vid, edges_ptr_[_].dst);
                    WriteMax(&max_vid, edges_ptr_[_].src);
                    WriteMax(&max_vid, edges_ptr_[_].dst);
                  }
                });

  Bitmap visited(max_vid);

  // Get number of vertices.
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, &visited, step, n_edges](auto w) {
                  for (EdgeIndex _ = w; _ < n_edges; _ += step) {
                    visited.SetBit(edges_ptr_[_].src);
                    visited.SetBit(edges_ptr_[_].dst);
                  }
                });

  edgelist_metadata_.num_edges = n_edges;
  edgelist_metadata_.num_vertices = visited.Count();
  edgelist_metadata_.max_vid = max_vid;
  edgelist_metadata_.min_vid = min_vid;
  if (localid2globalid == nullptr) {
    GenerateLocalID2GlobalID();
  } else {
    localid_to_globalid_ = localid2globalid;
  }
}

void Edges::WriteToBinary(const std::string &output_path) {
  if (!std::filesystem::exists(output_path))
    std::filesystem::create_directory(output_path);

  std::ofstream out_data_file(output_path + "edgelist.bin");
  std::ofstream out_localid2globalid_file(output_path + "localid2globalid.bin");
  std::ofstream out_meta_file(output_path + "meta.yaml");

  out_data_file.write(reinterpret_cast<char *>(edges_ptr_),
                      sizeof(Edge) * edgelist_metadata_.num_edges);

  out_localid2globalid_file.write(
      reinterpret_cast<char *>(localid_to_globalid_),
      sizeof(VertexID) * edgelist_metadata_.num_vertices);

  YAML::Node node;
  node["EdgelistBin"]["num_vertices"] = edgelist_metadata_.num_vertices;
  node["EdgelistBin"]["num_edges"] = edgelist_metadata_.num_edges;
  node["EdgelistBin"]["max_vid"] = edgelist_metadata_.max_vid;
  node["EdgelistBin"]["min_vid"] = edgelist_metadata_.min_vid;
  out_meta_file << node << std::endl;

  out_data_file.close();
  out_localid2globalid_file.close();
  out_meta_file.close();
}

void Edges::ReadFromBin(const std::string &input_path) {
  YAML::Node node = YAML::LoadFile(input_path + "meta.yaml");

  edgelist_metadata_ = {node["EdgelistBin"]["num_vertices"].as<VertexID>(),
                        node["EdgelistBin"]["num_edges"].as<EdgeIndex>(),
                        node["EdgelistBin"]["max_vid"].as<VertexID>()};

  edges_ptr_ =
      new sics::matrixgraph::core::data_structures::Edge[edgelist_metadata_
                                                             .num_edges]();

  std::ifstream in_file(input_path + "edgelist.bin");
  if (!in_file) {
    std::cout << "Open file failed: " + input_path + "edgelist.bin"
              << std::endl;
    exit(EXIT_FAILURE);
  }
  in_file.read(reinterpret_cast<char *>(edges_ptr_),
               sizeof(Edge) * edgelist_metadata_.num_edges);

  std::ifstream in_localid2globalid_file(input_path + "localid2globalid.bin");
  if (!in_localid2globalid_file) {
    std::cout << "Open file failed: " + input_path + "localid2globalid.bin"
              << std::endl;
    exit(EXIT_FAILURE);
  }
  in_file.read(reinterpret_cast<char *>(localid_to_globalid_),
               sizeof(VertexID) * edgelist_metadata_.num_vertices);
}

void Edges::ReadFromCSV(const std::string &filename, const std::string &sep,
                        bool compressed) {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  std::ifstream in_file(filename);

  in_file.seekg(0, std::ios::end);
  size_t length = in_file.tellg();
  in_file.seekg(0, std::ios::beg);

  char *buff = new char[length]();
  in_file.read(buff, length);
  std::string content(buff, length);

  EdgeIndex n_edges = count(content.begin(), content.end(), '\n');
  auto buffer_edges = new VertexID[n_edges * 2]();
  std::stringstream ss(content);
  delete[] buff;

  EdgeIndex index = 0;
  VertexID max_vid = 0, compressed_vid = 0;
  std::string line, vid_str;

  while (getline(ss, line, '\n')) {
    if (*line.c_str() == '\0')
      break;
    std::stringstream ss_line(line);
    while (getline(ss_line, vid_str, *sep.c_str())) {
      VertexID vid = stoll(vid_str);
      sics::matrixgraph::core::util::atomic::WriteMax(&max_vid, vid);
      buffer_edges[index++] = vid;
    }
  }
  content.clear();
  in_file.close();

  auto aligned_max_vid = (((max_vid + 1) >> 6) << 6) + 64;
  edges_ptr_ = new Edge[n_edges]();
  Bitmap bitmap(aligned_max_vid);

  auto vid_map = new VertexID[aligned_max_vid]();
  auto compressed_buffer_edges = new VertexID[n_edges * 2]();

  // Compute the mapping between origin vid to compressed vid.
  for (EdgeIndex index = 0; index < n_edges * 2; index++) {
    if (!bitmap.GetBit(buffer_edges[index])) {
      bitmap.SetBit(buffer_edges[index]);
      vid_map[buffer_edges[index]] = compressed_vid++;
    }
  }

  for (EdgeIndex i = 0; i < n_edges; i++) {
    edges_ptr_[i].src = buffer_edges[2 * i];
    edges_ptr_[i].dst = buffer_edges[2 * i + 1];
  }

  delete[] buffer_edges;
  delete[] vid_map;
  delete[] compressed_buffer_edges;

  // Compute metadata.
  edgelist_metadata_.num_edges = n_edges;
  edgelist_metadata_.num_vertices = bitmap.Count();
  edgelist_metadata_.max_vid = max_vid;
  if (compressed) {
    std::cout << "[Edges] Reading CSV with compressed ..." << std::endl;
    GenerateLocalID2GlobalID();
  } else {
    std::cout << "[Edges] Reading CSV without compressed ..." << std::endl;
    if (localid_to_globalid_ != nullptr)
      delete[] localid_to_globalid_;
    localid_to_globalid_ = new VertexID[edgelist_metadata_.num_vertices]();
    std::for_each(std::execution::par, worker.begin(), worker.end(),
                  [this, step](auto w) {
                    for (auto i = w; i < get_metadata().num_vertices;
                         i += step) {
                      std::cout << "->" << localid_to_globalid_[i] << " " << i
                                << std::endl;
                      localid_to_globalid_[i] = i;
                      std::cout << "->" << localid_to_globalid_[i] << " " << i
                                << std::endl;
                    }
                  });
  }
}

void Edges::GenerateLocalID2GlobalID() {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  VertexID *new_localid_to_globalid =
      new VertexID[edgelist_metadata_.num_vertices]();

  VertexID *vid_map = new VertexID[edgelist_metadata_.max_vid + 1]();

  VertexID compressed_vid = 0;

  Bitmap bitmap(edgelist_metadata_.max_vid);

  for (EdgeIndex index = 0; index < edgelist_metadata_.num_edges; index++) {
    auto e = get_edge_by_index(index);
    if (localid_to_globalid_ != nullptr) {
      e.src = localid_to_globalid_[e.src];
      e.dst = localid_to_globalid_[e.dst];
    }

    if (!bitmap.GetBit(e.src)) {
      bitmap.SetBit(e.src);
      vid_map[e.src] = compressed_vid++;
    }
    if (!bitmap.GetBit(e.dst)) {
      bitmap.SetBit(e.dst);
      vid_map[e.dst] = compressed_vid++;
    }
  }

  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, step, &vid_map, &new_localid_to_globalid](auto w) {
                  for (auto i = w; i < get_metadata().num_edges; i += step) {
                    auto e = get_edge_by_index(i);
                    if (localid_to_globalid_ != nullptr) {
                      e.src = localid_to_globalid_[e.src];
                      e.dst = localid_to_globalid_[e.dst];
                    }
                    new_localid_to_globalid[vid_map[e.src]] = e.src;
                    new_localid_to_globalid[vid_map[e.dst]] = e.dst;
                    edges_ptr_[i].src = vid_map[e.src];
                    edges_ptr_[i].dst = vid_map[e.dst];
                  }
                });
  delete[] vid_map;
  delete[] localid_to_globalid_;
  localid_to_globalid_ = new_localid_to_globalid;
}

void Edges::Compacted() {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  if (localid_to_globalid_ == nullptr) {
    localid_to_globalid_ = new VertexID[get_metadata().num_vertices]();
  } else {
    memset(localid_to_globalid_, 0,
           sizeof(VertexID) * get_metadata().num_vertices);
  }

  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, step](auto w) {
                  for (auto i = w; i < get_metadata().num_edges; i += step) {
                    auto e = get_edge_by_index(i);
                    localid_to_globalid_[e.src] = e.src;
                    localid_to_globalid_[e.dst] = e.dst;
                  }
                });
}

void Edges::Transpose() {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, step](auto w) {
                  for (auto i = w; i < get_metadata().num_edges; i += step) {
                    VertexID tmp = edges_ptr_[i].src;
                    edges_ptr_[i].src = edges_ptr_[i].dst; // swap src and dst
                    edges_ptr_[i].dst = tmp;
                  }
                });
}

void Edges::SortBySrc() {
  std::sort(std::execution::par, edges_ptr_,
            edges_ptr_ + edgelist_metadata_.num_edges);
}

void Edges::ShowGraph(EdgeIndex n_edges) const {
  std::cout << "[ShowGraph] n_edges:" << edgelist_metadata_.num_edges
            << ", n_vertices:" << edgelist_metadata_.num_vertices << std::endl;
  EdgeIndex min_n_edges = min(edgelist_metadata_.num_edges, n_edges);
  if (localid_to_globalid_ != nullptr) {
    for (EdgeIndex i = 0; i < min_n_edges; i++) {
      std::cout << localid_to_globalid_[edges_ptr_[i].src] << " "
                << localid_to_globalid_[edges_ptr_[i].dst] << std::endl;
    }
  } else {
    for (EdgeIndex i = 0; i < min_n_edges; i++) {
      std::cout << edges_ptr_[i].src << " " << edges_ptr_[i].dst << std::endl;
    }
  }
}

VertexID Edges::get_globalid_by_localid(VertexID localid) const {
  if (localid_to_globalid_ == nullptr)
    return localid;
  return localid_to_globalid_[localid];
}

void Edges::SetLocalIDToGlobalID(VertexID *localid_to_globalid) {
  if (localid_to_globalid_ != nullptr)
    delete[] localid_to_globalid;
  localid_to_globalid_ = localid_to_globalid;
}

} // namespace data_structures
} // namespace core
} // namespace matrixgraph
} // namespace sics