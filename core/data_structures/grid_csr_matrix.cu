#include "hip/hip_runtime.h"
#include "core/data_structures/grid_csr_matrix.cuh"

#include <algorithm>
#include <cmath>
#include <execution>

namespace sics {
namespace matrixgraph {
namespace core {
namespace data_structures {

using GraphID = sics::matrixgraph::core::common::GraphID;
using VertexID = sics::matrixgraph::core::common::VertexID;

GridCSRMatrix::GridCSRMatrix(const GridGraphMetadata &metadata) {
  metadata_ = metadata;
  tiled_matrix_vec_.resize(pow(metadata_.n_chunks, 2));

  std::generate(std::execution::par, tiled_matrix_vec_.begin(),
                tiled_matrix_vec_.end(), []() { return new CSRTiledMatrix(); });
}

void GridCSRMatrix::Print() const {
  std::cout << "[GridCSRMatrix Print] " << metadata_.n_chunks << "x"
            << metadata_.n_chunks << std::endl;
  for (GraphID x = 0; x < metadata_.n_chunks; x++) {
    for (GraphID y = 0; y < metadata_.n_chunks; y++) {
      if (tiled_matrix_vec_[x * metadata_.n_chunks + y]
              ->GetMetadata()
              .n_nz_tile == 0) {
        continue;
      }
      std::cout << "Block (" << x << "," << y << ")" << std::endl;
      tiled_matrix_vec_[x * metadata_.n_chunks + y]->Print();
    }
  }
}

} // namespace data_structures
} // namespace core
} // namespace matrixgraph
} // namespace sics