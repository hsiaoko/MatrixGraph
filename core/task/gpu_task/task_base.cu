#include "core/task/gpu_task/task_base.cuh"
#include <hip/hip_runtime.h>

namespace sics {
namespace matrixgraph {
namespace core {
namespace task {

// hipStream_t TaskBase::GetStream(size_t task_id) {
//   std::lock_guard<std::mutex> lock(streams_mtx_);
//   auto iter = streams_by_task_id_.find(task_id);
//   if (iter == streams_by_task_id_.end()) {
//     // If stream doesn't exist create a new CUDA stream
//     hipStream_t* p_stream = new hipStream_t;
//     hipStreamCreate(p_stream);
//     streams_by_task_id_.insert(std::make_pair(task_id, p_stream));
//     return *p_stream;
//   } else {
//     // Return the existing stream
//     return *(iter->second);
//   }
// }
//
// bool TaskBase::IsTaskFinished(size_t task_id) {
//   auto iter = streams_by_task_id_.find(task_id);
//   if (iter == streams_by_task_id_.end()) {
//     return false;
//   } else {
//     hipError_t err = hipStreamQuery(*iter->second);
//     if (err == hipSuccess)
//       return true;
//     else if (err == hipErrorNotReady)
//       return false;
//   }
// }
//
// void TaskBase::ReleaseStream(size_t task_id) {
//   std::lock_guard<std::mutex> lock(streams_mtx_);
//   // Find the stream for the task
//   auto iter = streams_by_task_id_.find(task_id);
//   if (iter != streams_by_task_id_.end()) {
//     // If stream exist destroy the stream
//     hipStreamDestroy(*iter->second);
//     // Erase the stream from the map
//     streams_by_task_id_.erase(iter);
//   }
// }

}  // namespace task
}  // namespace core
}  // namespace matrixgraph
}  // namespace sics
