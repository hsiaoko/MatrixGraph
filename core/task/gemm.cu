#include "hip/hip_runtime.h"
#include "core/task/gemm.cuh"

#include <ctime>
#include <hip/hip_runtime.h>
#include <execution>
#include <iostream>
#include <mutex>
#include <thread>
#include <unordered_map>

#include "core/common/host_algorithms.cuh"
#include "core/data_structures/device_buffer.cuh"
#include "core/data_structures/host_buffer.cuh"
#include "core/data_structures/metadata.h"
#include "core/data_structures/unified_buffer.cuh"
#include "core/io/grid_csr_tiled_matrix_io.cuh"
#include "core/task/kernel/matrix_operations.cuh"
#include "core/util/atomic.h"
#include "core/util/bitmap.h"
#include "core/util/bitmap_no_ownership.h"

namespace sics {
namespace matrixgraph {
namespace core {
namespace task {

using sics::matrixgraph::core::data_structures::GridCSRTiledMatrix;
using sics::matrixgraph::core::io::GridCSRTiledMatrixIO;
using GirdTiledMatrix =
    sics::matrixgraph::core::data_structures::GridCSRTiledMatrix;
using DeviceOwnedBufferUint64 =
    sics::matrixgraph::core::data_structures::DeviceOwnedBuffer<uint64_t>;
using DeviceOwnedBufferUint32 =
    sics::matrixgraph::core::data_structures::DeviceOwnedBuffer<uint32_t>;
using DeviceOwnedBufferUint8 =
    sics::matrixgraph::core::data_structures::DeviceOwnedBuffer<uint8_t>;
using UnifiedOwnedBufferUint32 =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<uint32_t>;
using UnifiedOwnedBufferUint64 =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<uint64_t>;
using UnifiedOwnedBufferUint8 =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<uint8_t>;
using BufferUint64 = sics::matrixgraph::core::data_structures::Buffer<uint64_t>;
using BufferUint8 = sics::matrixgraph::core::data_structures::Buffer<uint8_t>;
using BufferUint32 = sics::matrixgraph::core::data_structures::Buffer<uint32_t>;
using DeviceOwnedBufferUint64 =
    sics::matrixgraph::core::data_structures::DeviceOwnedBuffer<uint64_t>;
using MatrixOperationsKernelWrapper =
    sics::matrixgraph::core::task::kernel::MatrixOperationsKernelWrapper;
using Bitmap = sics::matrixgraph::core::util::Bitmap;
using GPUBitmap = sics::matrixgraph::core::util::GPUBitmap;
using BitmapNoOwnerShip = sics::matrixgraph::core::util::BitmapNoOwnerShip;
using sics::matrixgraph::core::util::atomic::WriteAdd;
using TiledMatrixMetadata =
    sics::matrixgraph::core::data_structures::TiledMatrixMetadata;

// CUDA kernel to add elements of two arrays
__host__ void GEMM::LoadData() {
  std::cout << "[GEMM] LoadData()" << std::endl;
  GridCSRTiledMatrixIO grid_csr_tiled_matrix_io;

  grid_csr_tiled_matrix_io.Read(input_path_, &A_);
  grid_csr_tiled_matrix_io.Read(input_path_transposed_, &B_);
  A_->Print();
  B_->Print();
  C_ = new GridCSRTiledMatrix(A_->get_metadata());
}

__host__ void GEMM::InitC() {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::mutex mtx;
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  auto block_a = A_->GetTiledMatrixPtrByIdx(0);
  VertexID n_strips = block_a->GetMetadata().n_strips;
  VertexID tile_size = block_a->GetMetadata().tile_size;

  std::cout << "[InitResultMatrix]"
            << " Start - n_strips: " << n_strips << ", tile_size: " << tile_size
            << std::endl;

  VertexID M = A_->get_metadata().n_chunks;
  VertexID K = A_->get_metadata().n_chunks;
  VertexID N = B_->get_metadata().n_chunks;

  VertexID n_chunks = A_->get_metadata().n_chunks;

  std::vector<BufferUint64> buffers_matrix_a;
  std::vector<BufferUint64> buffers_matrix_b;
  std::vector<BufferUint64> buffers_matrix_c;
  buffers_matrix_a.resize(M * K);
  buffers_matrix_b.resize(N * K);
  buffers_matrix_c.resize(M * N);

  std::vector<BufferUint32> tile_offset_row;
  std::vector<BufferUint32> tile_count_row;
  std::vector<BufferUint32> tile_row_idx;
  std::vector<BufferUint32> tile_col_idx;
  tile_offset_row.resize(M * N);
  tile_count_row.resize(M * N);
  tile_row_idx.resize(M * N);
  tile_col_idx.resize(M * N);

  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, M, N, K, step, n_strips, &tile_count_row](auto w) {
                  for (VertexID i = w; i < M; i += step) {
                    for (VertexID j = 0; j < N; j++) {
                      tile_count_row[i * N + j].data =
                          new VertexID[n_strips + 1]();
                      tile_count_row[i * N + j].size =
                          sizeof(VertexID) * (n_strips + 1);
                    }
                  }
                });

  std::vector<DeviceOwnedBufferUint64> device_owned_buffers_matrix_a;
  std::vector<DeviceOwnedBufferUint64> device_owned_buffers_matrix_b;
  std::vector<DeviceOwnedBufferUint64> device_owned_buffers_matrix_c;
  device_owned_buffers_matrix_a.resize(M * K);
  device_owned_buffers_matrix_b.resize(K * N);
  device_owned_buffers_matrix_c.resize(M * N);

  std::vector<DeviceOwnedBufferUint32> device_owned_tile_offset_row;
  std::vector<DeviceOwnedBufferUint32> device_owned_tile_count_row;
  std::vector<DeviceOwnedBufferUint32> device_owned_tile_row_idx;
  std::vector<DeviceOwnedBufferUint32> device_owned_tile_col_idx;
  device_owned_tile_offset_row.resize(M * N);
  device_owned_tile_count_row.resize(M * N);
  device_owned_tile_row_idx.resize(M * N);
  device_owned_tile_col_idx.resize(M * N);

  std::vector<hipStream_t> p_streams_vec;
  p_streams_vec.resize(M * N);
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, M, N, K, step, &p_streams_vec, &mtx](auto w) {
                  for (VertexID i = w; i < N; i += step) {
                    for (VertexID j = 0; j < M; j++) {
                      hipSetDevice(common::hash_function(i * N + j) % 4);
                      hipStreamCreate(&p_streams_vec[i * N + j]);
                    }
                  }
                });

  // Step 1 compute layout_matrix for each block of matrix C.
  std::cout << "[InitResultMatrix] Computing layout matrix for each block ..."
            << std::endl;
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &device_owned_buffers_matrix_a,
       &device_owned_buffers_matrix_b, &device_owned_buffers_matrix_c,
       &p_streams_vec, &buffers_matrix_a, &buffers_matrix_b, &buffers_matrix_c,
       n_strips, n_chunks, tile_size, &mtx](auto w) {
        for (VertexID k = w; k < K; k += step) {
          for (VertexID i = 0; i < M; i++) {
            for (VertexID j = 0; j < N; j++) {

              auto block_a = A_->GetTiledMatrixPtrByIdx(i * K + k);
              auto block_b = B_->GetTiledMatrixPtrByIdx(j * K + k);

              if (block_a->GetMetadata().n_nz_tile == 0)
                continue;
              if (block_b->GetMetadata().n_nz_tile == 0)
                continue;

              hipSetDevice(common::hash_function(i * N + j) % 4);
              hipStream_t &p_stream = p_streams_vec[i * N + j];

              {
                std::lock_guard<std::mutex> lock(mtx);
                if (device_owned_buffers_matrix_c[i * N + j].GetPtr() ==
                    nullptr) {
                  buffers_matrix_c[i * N + j].size =
                      sizeof(uint64_t) * (WORD_OFFSET(n_strips * n_strips) + 1);
                  hipHostAlloc(&buffers_matrix_c[i * N + j].data,
                                buffers_matrix_c[i * N + j].size,
                                hipHostMallocDefault);

                  device_owned_buffers_matrix_c[i * N + j].Init(
                      buffers_matrix_c[i * N + j].GetSize(), p_stream);
                }
              }

              auto &matrix_a_buf = buffers_matrix_a[i * K + k];
              matrix_a_buf.data =
                  (uint64_t *)block_a->GetNzTileBitmapPtr()->data();
              matrix_a_buf.size =
                  block_a->GetNzTileBitmapPtr()->GetBufferSize();

              auto &matrix_b_buf = buffers_matrix_b[j * K + k];
              matrix_b_buf.data =
                  (uint64_t *)block_b->GetNzTileBitmapPtr()->data();
              matrix_b_buf.size =
                  block_b->GetNzTileBitmapPtr()->GetBufferSize();

              device_owned_buffers_matrix_a[i * K + k].Init(matrix_a_buf,
                                                            p_stream);
              device_owned_buffers_matrix_b[j * K + k].Init(matrix_b_buf,
                                                            p_stream);

              MatrixOperationsKernelWrapper::MatrixBitAnd(
                  p_stream, device_owned_buffers_matrix_a[i * K + k],
                  device_owned_buffers_matrix_b[j * K + k],
                  &device_owned_buffers_matrix_c[i * N + j], n_strips, n_strips,
                  n_strips);
            }
          }
        }
      });

  // Step 3 compute Nonzero tile
  std::vector<BufferUint64> buffers_matrix_c_count;
  buffers_matrix_c_count.resize(M * N);
  std::vector<DeviceOwnedBufferUint64> device_owned_buffers_matrix_c_count;
  device_owned_buffers_matrix_c_count.resize(M * N);

  hipDeviceSynchronize();
  std::cout << "[InitResultMatrix] Counting nz tile ..." << std::endl;
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, M, N, K, step, &buffers_matrix_c, &mtx,
                 &device_owned_buffers_matrix_c_count, &p_streams_vec,
                 &device_owned_buffers_matrix_c, n_strips](auto w) {
                  for (VertexID i = w; i < M; i += step) {
                    for (VertexID j = 0; j < N; j++) {
                      if (device_owned_buffers_matrix_c[i * N + j].GetPtr() ==
                          nullptr) {
                        continue;
                      }
                      hipSetDevice(common::hash_function(i * N + j) % 4);
                      hipStream_t &p_stream = p_streams_vec[i * N + j];
                      device_owned_buffers_matrix_c_count[i * N + j].Init(
                          sizeof(uint64_t), p_stream);

                      MatrixOperationsKernelWrapper::MatrixBitCount(
                          p_stream, device_owned_buffers_matrix_c[i * N + j],
                          &device_owned_buffers_matrix_c_count[i * N + j],
                          n_strips * n_strips);
                    }
                  }
                });

  // Copy data back to the host.
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &device_owned_buffers_matrix_a,
       &device_owned_buffers_matrix_b, &device_owned_buffers_matrix_c_count,
       &device_owned_buffers_matrix_c, &p_streams_vec, &buffers_matrix_c,
       &buffers_matrix_c_count, n_strips, n_chunks, tile_size, &mtx](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID j = 0; j < N; j++) {
            if (device_owned_buffers_matrix_c[i * N + j].GetPtr() == nullptr) {
              continue;
            }
            hipSetDevice(common::hash_function(i * N + j) % 4);
            hipStream_t &p_stream = p_streams_vec[i * N + j];
            buffers_matrix_c_count[i * N + j].size = sizeof(uint64_t);
            hipHostAlloc(&buffers_matrix_c_count[i * N + j].data,
                          buffers_matrix_c_count[i * N + j].size,
                          hipHostMallocDefault);
            hipMemcpyAsync(
                buffers_matrix_c_count[i * N + j].GetPtr(),
                device_owned_buffers_matrix_c_count[i * N + j].GetPtr(),
                device_owned_buffers_matrix_c_count[i * N + j].GetSize(),
                hipMemcpyDeviceToHost, p_stream);
            hipMemcpyAsync(buffers_matrix_c[i * N + j].GetPtr(),
                            device_owned_buffers_matrix_c[i * N + j].GetPtr(),
                            device_owned_buffers_matrix_c[i * N + j].GetSize(),
                            hipMemcpyDeviceToHost, p_stream);
          }
        }
      });

  std::cout << "[InitResultMatrix] Allocating space for Matrix C ..."
            << std::endl;
  std::for_each(
      // std::execution::par,
      worker.begin(), worker.end(),
      [this, M, N, K, step, &device_owned_buffers_matrix_a,
       &device_owned_buffers_matrix_b, &device_owned_buffers_matrix_c_count,
       &device_owned_buffers_matrix_c, &p_streams_vec, &buffers_matrix_c_count,
       &buffers_matrix_c, n_strips, n_chunks, tile_size, &mtx](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID j = 0; j < N; j++) {
            if (device_owned_buffers_matrix_c[i * N + j].GetPtr() == nullptr) {
              continue;
            }
            VertexID n_nz_tile = *buffers_matrix_c_count[i * N + j].GetPtr();
            if (n_nz_tile == 0)
              continue;
            hipSetDevice(common::hash_function(i * N + j) % 4);
            hipStream_t &p_stream = p_streams_vec[i * N + j];

            auto *csr_tiled_matrix_ptr = C_->GetTiledMatrixPtrByIdx(i * N + j);

            TiledMatrixMetadata metadata{.n_strips = n_strips,
                                         .n_nz_tile = n_nz_tile,
                                         .tile_size = tile_size};
            csr_tiled_matrix_ptr->Init(
                metadata, new GPUBitmap(n_strips * tile_size,
                                        buffers_matrix_c[i * N + j].GetPtr()));
          }
        }
      });

  std::cout << "[InitResultMatrix] Initialize BitTiledMatrix metadata."
            << std::endl;
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &device_owned_buffers_matrix_c_count,
       &device_owned_buffers_matrix_c, &tile_count_row, &tile_row_idx,
       &tile_col_idx, &device_owned_tile_offset_row,
       &device_owned_tile_count_row, &device_owned_tile_row_idx,
       &device_owned_tile_col_idx,

       &p_streams_vec, &buffers_matrix_c_count, &buffers_matrix_c, n_strips,
       n_chunks, tile_size, &mtx](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID j = 0; j < N; j++) {
            if (device_owned_buffers_matrix_c[i * N + j].GetPtr() == nullptr) {
              continue;
            }
            VertexID n_nz_tile = *buffers_matrix_c_count[i * N + j].GetPtr();
            if (n_nz_tile == 0)
              continue;

            hipSetDevice(common::hash_function(i * N + j) % 4);
            hipStream_t &p_stream = p_streams_vec[i * N + j];
            auto *bit_tiled_matrix_ptr = C_->GetTiledMatrixPtrByIdx(i * N + j);

            tile_col_idx[i * N + j].data =
                bit_tiled_matrix_ptr->GetTileColIdxPtr();
            tile_col_idx[i * N + j].size =
                bit_tiled_matrix_ptr->GetMetadata().n_nz_tile *
                sizeof(VertexID);

            tile_row_idx[i * N + j].data =
                bit_tiled_matrix_ptr->GetTileRowIdxPtr();
            tile_row_idx[i * N + j].size =
                bit_tiled_matrix_ptr->GetMetadata().n_nz_tile *
                sizeof(VertexID);

            device_owned_tile_row_idx[i * N + j].Init(tile_row_idx[i * N + j],
                                                      p_stream);
            device_owned_tile_col_idx[i * N + j].Init(tile_col_idx[i * N + j],
                                                      p_stream);
            device_owned_tile_count_row[i * N + j].Init(
                sizeof(VertexID) * (n_strips + 1), p_stream);

            MatrixOperationsKernelWrapper::
                InitBitTiledMatrixMetadataByLayoutMatrix(
                    p_stream, device_owned_buffers_matrix_c[i * N + j],
                    &device_owned_tile_count_row[i * N + j],
                    &device_owned_tile_row_idx[i * N + j],
                    &device_owned_tile_col_idx[i * N + j],
                    bit_tiled_matrix_ptr->GetMetadata().tile_size);
          }
        }
      });
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &device_owned_buffers_matrix_c_count,
       &device_owned_buffers_matrix_c, &tile_count_row, &tile_row_idx,
       &tile_col_idx, &device_owned_tile_count_row, &device_owned_tile_row_idx,
       &device_owned_tile_col_idx,

       &p_streams_vec, &buffers_matrix_c_count, &buffers_matrix_c, n_strips,
       n_chunks, tile_size, &mtx](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID j = 0; j < N; j++) {
            if (device_owned_buffers_matrix_c[i * N + j].GetPtr() == nullptr) {
              continue;
            }
            VertexID n_nz_tile = *buffers_matrix_c_count[i * N + j].GetPtr();
            if (n_nz_tile == 0)
              continue;

            hipSetDevice(common::hash_function(i * N + j) % 4);
            hipStream_t &p_stream = p_streams_vec[i * N + j];

            hipMemcpyAsync(tile_count_row[i * N + j].GetPtr(),
                            device_owned_tile_count_row[i * N + j].GetPtr(),
                            device_owned_tile_count_row[i * N + j].GetSize(),
                            hipMemcpyDeviceToHost, p_stream);

            hipMemcpyAsync(tile_row_idx[i * N + j].GetPtr(),
                            device_owned_tile_row_idx[i * N + j].GetPtr(),
                            device_owned_tile_row_idx[i * N + j].GetSize(),
                            hipMemcpyDeviceToHost, p_stream);

            hipMemcpyAsync(tile_col_idx[i * N + j].GetPtr(),
                            device_owned_tile_col_idx[i * N + j].GetPtr(),
                            device_owned_tile_col_idx[i * N + j].GetSize(),
                            hipMemcpyDeviceToHost, p_stream);
          }
        }
      });
  hipDeviceSynchronize();
  std::cout << "[InitResultMatrix] Computing tile_offset_row for each "
               "TiledMatrix ..."
            << std::endl;
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &device_owned_buffers_matrix_c_count,
       &device_owned_buffers_matrix_c, &tile_offset_row, &tile_count_row,
       &tile_row_idx, &tile_col_idx, &device_owned_tile_offset_row,
       &device_owned_tile_row_idx, &device_owned_tile_col_idx,

       &p_streams_vec, &buffers_matrix_c_count, &buffers_matrix_c, n_strips,
       n_chunks, tile_size, &mtx](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID j = 0; j < N; j++) {
            if (device_owned_buffers_matrix_c[i * N + j].GetPtr() == nullptr) {
              continue;
            }
            VertexID n_nz_tile = *buffers_matrix_c_count[i * N + j].GetPtr();
            if (n_nz_tile == 0)
              continue;

            auto *csr_tiled_matrix_ptr = C_->GetTiledMatrixPtrByIdx(i * N + j);
            auto tile_offset_row = csr_tiled_matrix_ptr->GetTileOffsetRowPtr();

            for (int t = 0; t < csr_tiled_matrix_ptr->GetMetadata().n_strips;
                 t++) {
              tile_offset_row[t + 1] =
                  tile_offset_row[t] + tile_count_row[i * N + j].GetPtr()[t];
            }
          }
        }
      });

  std::cout << "[InitResultMatrix] Done!" << std::endl;
  std::for_each(tile_count_row.begin(), tile_count_row.end(),
                [](auto &d) { delete[] d.data; });
  std::for_each(p_streams_vec.begin(), p_streams_vec.end(),
                [](auto &s) { hipStreamDestroy(s); });
  std::for_each(buffers_matrix_c_count.begin(), buffers_matrix_c_count.end(),
                [](auto &d) { hipHostFree(d.data); });
}

__host__ void GEMM::FillTilesUnifiedMemory() {
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::mutex mtx;
  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  auto block_a = A_->GetTiledMatrixPtrByIdx(0);
  VertexID n_strips = block_a->GetMetadata().n_strips;
  VertexID tile_size = block_a->GetMetadata().tile_size;
  auto tile_buffer_size =
      sizeof(uint64_t) * std::max(1u, WORD_OFFSET(tile_size * tile_size));

  VertexID M = A_->get_metadata().n_chunks;
  VertexID K = A_->get_metadata().n_chunks;
  VertexID N = B_->get_metadata().n_chunks;
  std::vector<hipStream_t> p_streams_vec;
  p_streams_vec.resize(M * N);
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [this, M, N, K, step, &p_streams_vec, &mtx](auto w) {
                  for (VertexID i = w; i < N; i += step) {
                    for (VertexID j = 0; j < M; j++) {
                      hipSetDevice(common::hash_function(i * N + j) % 4);
                      hipStreamCreate(&p_streams_vec[i * N + j]);
                    }
                  }
                });

  std::cout << "[FillTiles]"
            << " Start - n_strips: " << n_strips << ", tile_size: " << tile_size
            << std::endl;

  std::vector<BufferUint64> layout_matrix_c;
  std::vector<BufferUint32> tile_offset_row_a;
  std::vector<BufferUint32> tile_offset_row_b;
  std::vector<BufferUint32> tile_offset_row_c;
  std::vector<BufferUint32> tile_row_idx_a;
  std::vector<BufferUint32> tile_row_idx_b;
  std::vector<BufferUint32> tile_row_idx_c;
  std::vector<BufferUint32> tile_col_idx_a;
  std::vector<BufferUint32> tile_col_idx_b;
  std::vector<BufferUint32> tile_col_idx_c;
  std::vector<BufferUint64> csr_offset_a;
  std::vector<BufferUint64> csr_offset_b;
  std::vector<BufferUint64> csr_offset_c;
  std::vector<BufferUint8> data_a;
  std::vector<BufferUint8> data_b;
  std::vector<BufferUint8> data_c;

  layout_matrix_c.resize(M * N);
  tile_offset_row_a.resize(M * K);
  tile_offset_row_b.resize(N * K);
  tile_offset_row_c.resize(M * N);
  tile_row_idx_a.resize(M * K);
  tile_row_idx_b.resize(N * K);
  tile_row_idx_c.resize(M * N);
  tile_col_idx_a.resize(M * K);
  tile_col_idx_b.resize(N * K);
  tile_col_idx_c.resize(M * N);
  csr_offset_a.resize(M * K);
  csr_offset_b.resize(N * K);
  csr_offset_c.resize(M * N);
  data_a.resize(M * K);
  data_b.resize(N * K);
  data_c.resize(M * N);

  std::vector<UnifiedOwnedBufferUint64> unified_layout_matrix_c;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_offset_row_a;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_offset_row_b;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_offset_row_c;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_row_idx_a;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_row_idx_b;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_row_idx_c;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_col_idx_a;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_col_idx_b;
  std::vector<UnifiedOwnedBufferUint32> unified_tile_col_idx_c;
  std::vector<UnifiedOwnedBufferUint64> unified_csr_offset_a;
  std::vector<UnifiedOwnedBufferUint64> unified_csr_offset_b;
  std::vector<UnifiedOwnedBufferUint64> unified_csr_offset_c;
  std::vector<UnifiedOwnedBufferUint8> unified_data_a;
  std::vector<UnifiedOwnedBufferUint8> unified_data_b;
  std::vector<UnifiedOwnedBufferUint8> unified_data_c;

  unified_layout_matrix_c.resize(M * N);
  unified_tile_offset_row_a.resize(M * K);
  unified_tile_offset_row_b.resize(N * K);
  unified_tile_offset_row_c.resize(M * N);
  unified_tile_row_idx_a.resize(M * K);
  unified_tile_row_idx_b.resize(N * K);
  unified_tile_row_idx_c.resize(M * N);
  unified_tile_col_idx_a.resize(M * K);
  unified_tile_col_idx_b.resize(N * K);
  unified_tile_col_idx_c.resize(M * N);
  unified_csr_offset_a.resize(M * K);
  unified_csr_offset_b.resize(N * K);
  unified_csr_offset_c.resize(M * N);
  unified_data_a.resize(M * K);
  unified_data_b.resize(N * K);
  unified_data_c.resize(M * N);

  std::cout << "[FillTiles] Initializing buffers for results ..." << std::endl;
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &p_streams_vec, tile_size, tile_buffer_size, &mtx,
       &layout_matrix_c, &tile_offset_row_c, &tile_row_idx_c, &tile_col_idx_c,
       &csr_offset_c, &data_c, &unified_layout_matrix_c,
       &unified_tile_offset_row_c, &unified_tile_row_idx_c,
       &unified_tile_col_idx_c, &unified_csr_offset_c,
       &unified_data_c](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID j = 0; j < N; j++) {
            auto block_c = C_->GetTiledMatrixPtrByIdx(i * N + j);
            if (block_c == nullptr)
              continue;
            if (block_c->GetMetadata().n_nz_tile == 0)
              continue;

            layout_matrix_c[i * N + j].data =
                (uint64_t *)block_c->GetNzTileBitmapPtr()->data();

            layout_matrix_c[i * N + j].size =
                block_c->GetNzTileBitmapPtr()->size();

            tile_offset_row_c[i * N + j].data = block_c->GetTileOffsetRowPtr();
            tile_offset_row_c[i * N + j].size =
                sizeof(VertexID) * (block_c->GetMetadata().n_strips + 1);

            tile_row_idx_c[i * N + j].data = block_c->GetTileRowIdxPtr();
            tile_row_idx_c[i * N + j].size =
                block_c->GetMetadata().n_nz_tile * sizeof(VertexID);

            tile_col_idx_c[i * N + j].data = block_c->GetTileColIdxPtr();
            tile_col_idx_c[i * N + j].size =
                block_c->GetMetadata().n_nz_tile * sizeof(VertexID);
            csr_offset_c[i * N + j].data = block_c->GetCSROffsetPtr();
            csr_offset_c[i * N + j].size =
                block_c->GetMetadata().n_nz_tile * sizeof(VertexID);

            data_c[i * N + j].data = block_c->GetDataPtr();
            data_c[i * N + j].size = block_c->GetDataBufferSize();

            {
              std::lock_guard<std::mutex> lock(mtx);
              unified_layout_matrix_c[i * N + j].Init(
                  layout_matrix_c[i * N + j]);

              unified_tile_offset_row_c[i * N + j].Init(
                  tile_offset_row_c[i * N + j]);

              unified_tile_row_idx_c[i * N + j].Init(tile_row_idx_c[i * N + j]);
              unified_tile_col_idx_c[i * N + j].Init(tile_col_idx_c[i * N + j]);
              unified_csr_offset_c[i * N + j].Init(csr_offset_c[i * N + j]);
              unified_data_c[i * N + j].Init(data_c[i * N + j]);
            }
          }
        }
      });

  // Init input Buffer for A_ and B, respectively.
  std::cout << "[FillTiles] Initializing input buffers for A and B."
            << std::endl;
  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &p_streams_vec, tile_size, tile_buffer_size, &mtx,
       &tile_offset_row_a, &tile_row_idx_a, &tile_col_idx_a, &csr_offset_a,
       &data_a, &unified_tile_offset_row_a, &unified_tile_row_idx_a,
       &unified_tile_col_idx_a, &unified_csr_offset_a,
       &unified_data_a](auto w) {
        for (VertexID i = w; i < M; i += step) {
          for (VertexID k = 0; k < K; k++) {
            auto block_a = A_->GetTiledMatrixPtrByIdx(i * K + k);
            if (block_a->GetMetadata().n_nz_tile == 0)
              continue;

            tile_offset_row_a[i * K + k].data = block_a->GetTileOffsetRowPtr();
            tile_offset_row_a[i * K + k].size =
                sizeof(VertexID) * (block_a->GetMetadata().n_strips + 1);

            tile_row_idx_a[i * K + k].data = block_a->GetTileRowIdxPtr();
            tile_row_idx_a[i * K + k].size =
                block_a->GetMetadata().n_nz_tile * sizeof(VertexID);

            tile_col_idx_a[i * K + k].data = block_a->GetTileColIdxPtr();
            tile_col_idx_a[i * K + k].size =
                block_a->GetMetadata().n_nz_tile * sizeof(VertexID);
            csr_offset_a[i * K + k].data = block_a->GetCSROffsetPtr();
            csr_offset_a[i * K + k].size =
                block_a->GetMetadata().n_nz_tile * sizeof(VertexID);

            data_a[i * K + k].data = block_a->GetDataPtr();
            data_a[i * K + k].size =
                tile_buffer_size * block_a->GetMetadata().n_nz_tile;

            {
              std::lock_guard<std::mutex> lock(mtx);
              unified_tile_offset_row_a[i * K + k].Init(
                  tile_offset_row_a[i * K + k]);
              unified_tile_row_idx_a[i * K + k].Init(tile_row_idx_a[i * K + k]);
              unified_tile_col_idx_a[i * K + k].Init(tile_col_idx_a[i * K + k]);
              unified_csr_offset_a[i * K + k].Init(csr_offset_a[i * K + k]);
              unified_data_a[i * K + k].Init(data_a[i * K + k]);
            }
          }
        }
      });

  std::for_each(
      std::execution::par, worker.begin(), worker.end(),
      [this, M, N, K, step, &p_streams_vec, tile_size, tile_buffer_size, &mtx,
       &tile_offset_row_b, &tile_row_idx_b, &tile_col_idx_b, &csr_offset_b,
       &data_b, &unified_tile_offset_row_b, &unified_tile_row_idx_b,
       &unified_tile_col_idx_b, &unified_csr_offset_b,
       &unified_data_b](auto w) {
        for (VertexID i = w; i < N; i += step) {
          for (VertexID k = 0; k < K; k++) {
            auto block_b = B_->GetTiledMatrixPtrByIdx(i * K + k);
            if (block_b->GetMetadata().n_nz_tile == 0)
              continue;

            tile_offset_row_b[i * K + k].data = block_b->GetTileOffsetRowPtr();
            tile_offset_row_b[i * K + k].size =
                sizeof(VertexID) * (block_b->GetMetadata().n_strips + 1);

            tile_row_idx_b[i * K + k].data = block_b->GetTileRowIdxPtr();
            tile_row_idx_b[i * K + k].size =
                block_b->GetMetadata().n_nz_tile * sizeof(VertexID);

            tile_col_idx_b[i * K + k].data = block_b->GetTileColIdxPtr();
            tile_col_idx_b[i * K + k].size =
                block_b->GetMetadata().n_nz_tile * sizeof(VertexID);
            csr_offset_b[i * K + k].data = block_b->GetCSROffsetPtr();
            csr_offset_b[i * K + k].size =
                block_b->GetMetadata().n_nz_tile * sizeof(VertexID);

            data_b[i * K + k].data = block_b->GetDataPtr();
            data_b[i * K + k].size =
                tile_buffer_size * block_b->GetMetadata().n_nz_tile;

            {
              std::lock_guard<std::mutex> lock(mtx);
              unified_tile_offset_row_b[i * K + k].Init(
                  tile_offset_row_b[i * K + k]);
              unified_tile_row_idx_b[i * K + k].Init(tile_row_idx_b[i * K + k]);
              unified_tile_col_idx_b[i * K + k].Init(tile_col_idx_b[i * K + k]);
              unified_data_b[i * K + k].Init(data_b[i * K + k]);
              unified_csr_offset_b[i * K + k].Init(csr_offset_b[i * K + k]);
            }
          }
        }
      });

  hipDeviceSynchronize();
  // Submit Kernel to fill edges into tiles.
  std::vector<int> work_load;
  work_load.resize(4);
  std::cout << "[FillTiles] Filling tiles ..." << std::endl;
  auto start_time_1 = std::chrono::system_clock::now();
  std::for_each(
      // std::execution::par,
      worker.begin(), worker.end(),
      [this, M, N, K, step, &p_streams_vec, tile_size, tile_buffer_size,
       n_strips, &mtx, &unified_layout_matrix_c, &unified_tile_offset_row_a,
       &unified_tile_row_idx_a, &unified_tile_col_idx_a, &unified_csr_offset_a,
       &unified_data_a, &unified_tile_offset_row_b, &unified_tile_row_idx_b,
       &unified_tile_col_idx_b, &unified_csr_offset_b, &unified_data_b,
       &unified_tile_offset_row_c, &unified_tile_row_idx_c,
       &unified_tile_col_idx_c, &unified_csr_offset_c, &unified_data_c,
       &work_load](auto w) {
        for (VertexID k = w; k < K; k += step) {
          for (VertexID i = 0; i < M; i++) {
            for (VertexID j = 0; j < N; j++) {
              auto block_a = A_->GetTiledMatrixPtrByIdx(i * K + k);
              auto block_b = B_->GetTiledMatrixPtrByIdx(j * K + k);
              auto block_c = C_->GetTiledMatrixPtrByIdx(i * N + j);

              if (block_c == nullptr)
                continue;
              if (block_b == nullptr)
                continue;
              if (block_a == nullptr)
                continue;

              if (block_a->GetMetadata().n_nz_tile == 0)
                continue;
              if (block_b->GetMetadata().n_nz_tile == 0)
                continue;
              if (block_c->GetMetadata().n_nz_tile == 0)
                continue;

              std::cout << "- A - " << std::endl;
              block_a->Print();
              std::cout << "- B - " << std::endl;
              block_b->Print();
              std::cout << "- C -" << std::endl;

              block_c->Print();

              for (int _ = 0; _ < block_a->GetMetadata().n_nz_tile; _++) {
                std::cout << unified_csr_offset_a[i * K + k].GetPtr()[_] << " ";
              }
              std::cout << std::endl;

              for (int _ = 0; _ < block_b->GetMetadata().n_nz_tile; _++) {
                std::cout << unified_csr_offset_b[j * K + k].GetPtr()[_] << " ";
              }
              std::cout << std::endl;

              for (int _ = 0; _ < block_c->GetMetadata().n_nz_tile; _++) {
                std::cout << unified_csr_offset_c[i * N + k].GetPtr()[_] << " ";
              }
              std::cout << std::endl;

              WriteAdd(&work_load[common::hash_function(i * N + j) % 4], 1);
              {
                std::lock_guard<std::mutex> lock(mtx);
                hipSetDevice(common::hash_function(i * N + j) % 4);
                hipStream_t &p_stream = p_streams_vec[i * N + j];
                MatrixOperationsKernelWrapper::FillCSRTiles(
                    p_stream, tile_size, n_strips,
                    block_a->GetMetadata().n_nz_tile,
                    block_b->GetMetadata().n_nz_tile,
                    block_c->GetMetadata().n_nz_tile,
                    unified_layout_matrix_c[i * N + j],
                    unified_tile_offset_row_a[i * K + k],
                    unified_tile_offset_row_b[j * K + k],
                    unified_tile_offset_row_c[i * N + j],
                    unified_tile_row_idx_a[i * K + k],
                    unified_tile_row_idx_b[j * K + k],
                    unified_tile_row_idx_c[i * N + j],
                    unified_tile_col_idx_a[i * K + k],
                    unified_tile_col_idx_b[j * K + k],
                    unified_tile_col_idx_c[i * N + j],
                    unified_csr_offset_a[i * K + k],
                    unified_csr_offset_b[j * K + k],
                    unified_csr_offset_c[i * N + k], unified_data_a[i * K + k],
                    unified_data_b[j * K + k], &unified_data_c[i * N + j]);
              }
              while (1)
                ;
            }
          }
        }
      });

  std::cout << "END" << std::endl;
}

__host__ void GEMM::FillTiles() {}

__host__ void GEMM::Count(const GridCSRTiledMatrix &G) {}

__host__ void GEMM::Run() {

  auto start_time_0 = std::chrono::system_clock::now();

  InitC();

  auto start_time_1 = std::chrono::system_clock::now();

  std::cout << "[GEMM] Run Step1 InitResultMatrix() elapsed: "
            << std::chrono::duration_cast<std::chrono::microseconds>(
                   start_time_1 - start_time_0)
                       .count() /
                   (double)CLOCKS_PER_SEC
            << std::endl;

  FillTilesUnifiedMemory();
  auto start_time_2 = std::chrono::system_clock::now();
  std::cout << "[GEMM] Run Step2 FillTiles() elapsed:"
            << std::chrono::duration_cast<std::chrono::microseconds>(
                   start_time_2 - start_time_1)
                       .count() /
                   (double)CLOCKS_PER_SEC
            << std::endl;

  auto start_time_3 = std::chrono::system_clock::now();
  std::cout << "[GEMM] Run Step3 Count() elapsed:"
            << std::chrono::duration_cast<std::chrono::microseconds>(
                   start_time_3 - start_time_2)
                       .count() /
                   (double)CLOCKS_PER_SEC
            << std::endl;
}

} // namespace task
} // namespace core
} // namespace matrixgraph
} // namespace sics