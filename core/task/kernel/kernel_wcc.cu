#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

#include "core/common/consts.h"
#include "core/common/types.h"
#include "core/task/kernel/data_structures/kernel_bitmap.cuh"
#include "core/task/kernel/data_structures/kernel_bitmap_no_ownership.cuh"
#include "core/task/kernel/data_structures/mini_kernel_bitmap.cuh"
#include "core/task/kernel/kernel_wcc.cuh"
#include "core/util/bitmap_no_ownership.h"

namespace sics {
namespace matrixgraph {
namespace core {
namespace task {
namespace kernel {

using EdgeIndex = sics::matrixgraph::core::common::EdgeIndex;
using VertexLabel = sics::matrixgraph::core::common::VertexLabel;
using sics::matrixgraph::core::common::kMaxNumCandidatesPerThread;
using VertexID = sics::matrixgraph::core::common::VertexID;
using VertexID = sics::matrixgraph::core::common::VertexID;
using sics::matrixgraph::core::common::kBlockDim;
using sics::matrixgraph::core::common::kGridDim;
using sics::matrixgraph::core::common::kMaxNumWeft;
using sics::matrixgraph::core::common::kMaxVertexID;
using sics::matrixgraph::core::task::kernel::HostKernelBitmap;
using sics::matrixgraph::core::task::kernel::HostMiniKernelBitmap;
using sics::matrixgraph::core::task::kernel::KernelBitmap;
using sics::matrixgraph::core::task::kernel::KernelBitmapNoOwnership;
using sics::matrixgraph::core::task::kernel::MiniKernelBitmap;
using sics::matrixgraph::core::util::BitmapNoOwnerShip;

struct ParametersWCC {
  VertexID n_vertices_g;
  EdgeIndex n_edges_g;
  uint8_t* data_g;
  VertexLabel* v_label_g;
  VertexID* in_active_vertices;
  VertexID* out_active_vertices;
  VertexID* in_active_vertices_offset;
  VertexID* out_active_vertices_offset;
  uint64_t* in_visited_bitmap_data;
  uint64_t* out_visited_bitmap_data;
};

static __global__ void InitKernel(ParametersWCC params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;
  for (VertexID v_idx = tid; v_idx < params.n_vertices_g; v_idx += step) {
    params.v_label_g[v_idx] = v_idx;
    params.in_active_vertices[v_idx] = v_idx;
  }
}

static __global__ void HashMinKernel(ParametersWCC params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  VertexID* globalid_g = (VertexID*)(params.data_g);
  VertexID* in_degree_g = globalid_g + params.n_vertices_g;
  VertexID* out_degree_g = in_degree_g + params.n_vertices_g;
  EdgeIndex* in_offset_g = (EdgeIndex*)(out_degree_g + params.n_vertices_g);
  EdgeIndex* out_offset_g = (EdgeIndex*)(in_offset_g + params.n_vertices_g + 1);
  EdgeIndex* in_edges_g = (EdgeIndex*)(out_offset_g + params.n_vertices_g + 1);
  VertexID* out_edges_g = in_edges_g + params.n_edges_g;
  VertexID* edges_globalid_by_localid_g = out_edges_g + params.n_edges_g;

  KernelBitmapNoOwnership in_visited(params.n_vertices_g,
                                     params.in_visited_bitmap_data);
  KernelBitmapNoOwnership out_visited(params.n_vertices_g,
                                      params.out_visited_bitmap_data);

  for (VertexID v_idx = tid; v_idx < params.n_vertices_g; v_idx += step) {
    if (!in_visited.GetBit(v_idx)) continue;
    EdgeIndex v_offset_base = out_offset_g[v_idx];

    VertexLabel v_label = params.v_label_g[v_idx];
    for (VertexID nbr_v_idx = 0; nbr_v_idx < out_degree_g[v_idx]; nbr_v_idx++) {
      VertexID nbr_v = out_edges_g[v_offset_base + nbr_v_idx];

      // if (nbr_v >= 65608366) {
      //   printf("%d/ 65608366\n", nbr_v);
      // }

      VertexLabel label_nbr_v = *(params.v_label_g + nbr_v);

      if (label_nbr_v > v_label) {
        atomicMin(params.v_label_g + nbr_v, v_label);
        out_visited.SetBit(nbr_v);
      }
    }
  }
}

static __global__ void HashMinKernelActiveVertices(ParametersWCC params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;
  VertexID* globalid_g = (VertexID*)(params.data_g);
  VertexID* in_degree_g = globalid_g + params.n_vertices_g;
  VertexID* out_degree_g = in_degree_g + params.n_vertices_g;
  EdgeIndex* in_offset_g = (EdgeIndex*)(out_degree_g + params.n_vertices_g);
  EdgeIndex* out_offset_g = (EdgeIndex*)(in_offset_g + params.n_vertices_g + 1);
  EdgeIndex* in_edges_g = (EdgeIndex*)(out_offset_g + params.n_vertices_g + 1);
  VertexID* out_edges_g = in_edges_g + params.n_edges_g;
  VertexID* edges_globalid_by_localid_g = out_edges_g + params.n_edges_g;

  KernelBitmapNoOwnership in_visited(params.n_vertices_g,
                                     params.in_visited_bitmap_data);
  KernelBitmapNoOwnership out_visited(params.n_vertices_g,
                                      params.out_visited_bitmap_data);
  for (VertexID offset = tid; offset < *(params.in_active_vertices_offset);
       offset += step) {
    auto v_idx = params.in_active_vertices[offset];
    EdgeIndex v_offset_base = in_offset_g[v_idx];

    VertexLabel v_label = params.v_label_g[v_idx];
    VertexLabel min_label = v_label;

    for (VertexID nbr_v_idx = 0; nbr_v_idx < in_degree_g[v_idx]; nbr_v_idx++) {
      VertexID nbr_v = in_edges_g[v_offset_base + nbr_v_idx];

      VertexLabel label_nbr_v = *(params.v_label_g + nbr_v);

      if (label_nbr_v < v_label) {
        min_label = label_nbr_v;
      }
    }
    if (min_label < v_label) {
      atomicMin(params.v_label_g + v_idx, min_label);
      out_visited.SetBit(v_idx);
      auto offset = atomicAdd(params.out_active_vertices_offset, VertexID(1));
      // params.out_active_vertices[offset] = v_idx;
      //  printf("%d->%d ", label_nbr_v, v_label);
      //  printf("active: %d\n", params.out_active_vertices[offset]);
    }
  }
}

void WCCKernelWrapper::WCC(
    const hipStream_t& stream, VertexID n_vertices_g, EdgeIndex n_edges_g,
    const data_structures::UnifiedOwnedBuffer<uint8_t>& data_g,
    const data_structures::UnifiedOwnedBuffer<VertexLabel>& v_label_g) {
  dim3 dimBlock(kBlockDim);
  dim3 dimGrid(kGridDim);
  // dim3 dimBlock(1);
  // dim3 dimGrid(1);

  // The default heap size is 8M.
  hipDeviceSetLimit(hipLimitMallocHeapSize, 8388608 * 128);

  // Initialize.
  uint64_t* in_visited_bitmap_data;
  CUDA_CHECK(hipMallocManaged(
      &in_visited_bitmap_data,
      sizeof(uint64_t) * (KERNEL_WORD_OFFSET(n_vertices_g) + 1)));
  uint64_t* out_visited_bitmap_data;
  CUDA_CHECK(hipMallocManaged(
      &out_visited_bitmap_data,
      sizeof(uint64_t) * (KERNEL_WORD_OFFSET(n_vertices_g) + 1)));
  VertexID* in_active_vertices;
  VertexID* out_active_vertices;
  VertexID* in_active_vertices_offset;
  VertexID* out_active_vertices_offset;
  CUDA_CHECK(
      hipMallocManaged(&in_active_vertices, sizeof(VertexID) * n_vertices_g));
  CUDA_CHECK(
      hipMallocManaged(&out_active_vertices, sizeof(VertexID) * n_vertices_g));
  CUDA_CHECK(hipMallocManaged(&in_active_vertices_offset, sizeof(VertexID)));
  CUDA_CHECK(hipMallocManaged(&out_active_vertices_offset, sizeof(VertexID)));
  *in_active_vertices_offset = n_vertices_g;
  *out_active_vertices_offset = 0;

  BitmapNoOwnerShip out_visited(n_vertices_g, out_visited_bitmap_data);
  BitmapNoOwnerShip in_visited(n_vertices_g, in_visited_bitmap_data);

  ParametersWCC params{.n_vertices_g = n_vertices_g,
                       .n_edges_g = n_edges_g,
                       .data_g = data_g.GetPtr(),
                       .v_label_g = v_label_g.GetPtr(),
                       .in_active_vertices = in_active_vertices,
                       .out_active_vertices = out_active_vertices,
                       .in_active_vertices_offset = in_active_vertices_offset,
                       .out_active_vertices_offset = out_active_vertices_offset,
                       .in_visited_bitmap_data = in_visited.data(),
                       .out_visited_bitmap_data = out_visited.data()};

  InitKernel<<<dimGrid, dimBlock, 0, stream>>>(params);

  hipStreamSynchronize(stream);

  in_visited.Fill();
  size_t round = 0;
  auto time1 = std::chrono::system_clock::now();
  while (!in_visited.IsEmpty()) {
    ParametersWCC params{
        .n_vertices_g = n_vertices_g,
        .n_edges_g = n_edges_g,
        .data_g = data_g.GetPtr(),
        .v_label_g = v_label_g.GetPtr(),
        .in_active_vertices = in_active_vertices,
        .out_active_vertices = out_active_vertices,
        .in_active_vertices_offset = in_active_vertices_offset,
        .out_active_vertices_offset = out_active_vertices_offset,
        .in_visited_bitmap_data = in_visited.data(),
        .out_visited_bitmap_data = out_visited.data()};

    // std::cout << "Round " << round++
    //           << " Active vertices: " << in_visited.Count() << std::endl;
    std::cout << "Round " << round++
              << " Active vertices: " << *(in_active_vertices_offset)
              << std::endl;
    // HashMinKernel<<<dimGrid, dimBlock, 0, stream>>>(params);
    HashMinKernelActiveVertices<<<dimGrid, dimBlock, 0, stream>>>(params);
    hipStreamSynchronize(stream);

    std::swap(in_visited, out_visited);
    std::swap(in_active_vertices, out_active_vertices);
    std::swap(in_active_vertices_offset, out_active_vertices_offset);
    hipMemset(out_active_vertices, 0,
               sizeof(VertexID) * *(out_active_vertices_offset));
    *(out_active_vertices_offset) = 0;
    out_visited.Clear();
  }
  auto time2 = std::chrono::system_clock::now();

  for (int i = 0; i < 5; i++) {
    std::cout << v_label_g.GetPtr()[i] << " ";
  }
  std::cout << "[WCC]:"
            << std::chrono::duration_cast<std::chrono::microseconds>(time2 -
                                                                     time1)
                       .count() /
                   (double)CLOCKS_PER_SEC
            << "\n\t HashMin:"
            << std::chrono::duration_cast<std::chrono::microseconds>(time2 -
                                                                     time1)
                       .count() /
                   (double)CLOCKS_PER_SEC
            << std::endl;

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    CUDA_CHECK(err);
  }
  hipFree(in_visited_bitmap_data);
  hipFree(out_visited_bitmap_data);
  hipFree(in_active_vertices);
  hipFree(out_active_vertices);
  hipFree(in_active_vertices_offset);
  hipFree(out_active_vertices_offset);
}

}  // namespace kernel
}  // namespace task
}  // namespace core
}  // namespace matrixgraph
}  // namespace sics
