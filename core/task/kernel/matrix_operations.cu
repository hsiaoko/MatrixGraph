#include "hip/hip_runtime.h"
#include "core/task/kernel/matrix_operations.cuh"

#include <hip/hip_runtime.h>
#include <iostream>

namespace sics {
namespace matrixgraph {
namespace core {
namespace task {
namespace kernel {

#define WORD_OFFSET(i) (i >> 6)
#define BIT_OFFSET(i) (i & 0x3f)
#define min(a, b) ((a) < (b) ? (a) : (b))
#define max(a, b) ((a) > (b) ? (a) : (b))

static const uint32_t kProcessorWordSize = 64;

struct ParametersForMatrixBitAnd {
  uint64_t *matrix_a;
  uint64_t *matrix_b;
  uint64_t *matrix_c;
  uint64_t m;
  uint64_t k;
  uint64_t n;
};

struct ParametersForMatrixBitCount {
  unsigned long long *data;
  unsigned long long *count;
  unsigned long long size;
};

struct ParametersInitBitTiledMatrix {
  unsigned long long *layout_matrix;
  unsigned *tile_offset_row;
  unsigned *tile_row_idx;
  unsigned *tile_col_idx;
  unsigned long tile_size;
};

__device__ static inline uint64_t get_bit(uint64_t *data, size_t i,
                                          size_t size) {
  if (i > size)
    return 0;
  return data[WORD_OFFSET(i)] & (1ull << BIT_OFFSET(i));
}

__device__ static inline bool drop() { return false; }

__device__ static inline uint64_t
get_aligned_k_bits(uint64_t *data, uint64_t start, uint64_t end) {
  uint64_t start_word = WORD_OFFSET(start);
  uint64_t end_word = WORD_OFFSET(end);

  uint64_t start_bit = BIT_OFFSET(start);
  uint64_t end_bit = BIT_OFFSET(end);

  uint64_t result = 0;
  uint64_t mask = ((1ull << (end_bit - start_bit)) - 1) << start_bit;

  result = (data[start_word] & mask) >> start_bit;
  return result;
}

__device__ static inline uint64_t set_bit(unsigned long long *data,
                                          unsigned long long i) {
  atomicOr(data + WORD_OFFSET(i), (1ull << BIT_OFFSET(i)));
  // *(data + WORD_OFFSET(i)) |= (1ull << BIT_OFFSET(i));
}

__device__ static inline bool get_bit(uint64_t *data, uint64_t i) {
  return data[WORD_OFFSET(i)] & (1ull << BIT_OFFSET(i));
}

__device__ static inline size_t pre_element_count(const uint64_t *data,
                                                  uint64_t idx) {

  size_t count = 0;
  size_t bm_size = WORD_OFFSET(idx);
  size_t idx_offset = WORD_OFFSET(idx);
  size_t idx_bit_offset = BIT_OFFSET(idx);

  for (size_t i = 0; i <= bm_size; i++) {
    uint64_t x = 0;
    if (i == idx_offset) {
      uint64_t mask = (1ul << idx_bit_offset) - 1;
      x = data[i] & mask;
    } else {
      x = data[i];
    }
    x = (x & (0x5555555555555555)) + ((x >> 1) & (0x5555555555555555));
    x = (x & (0x3333333333333333)) + ((x >> 2) & (0x3333333333333333));
    x = (x & (0x0f0f0f0f0f0f0f0f)) + ((x >> 4) & (0x0f0f0f0f0f0f0f0f));
    x = (x & (0x00ff00ff00ff00ff)) + ((x >> 8) & (0x00ff00ff00ff00ff));
    x = (x & (0x0000ffff0000ffff)) + ((x >> 16) & (0x0000ffff0000ffff));
    x = (x & (0x00000000ffffffff)) + ((x >> 32) & (0x00000000ffffffff));
    count += (size_t)x;
  }

  return count;
};

static __global__ void matrix_and_kernel(ParametersForMatrixBitAnd params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int row_a = tid; row_a < params.m; row_a += step) {
    for (unsigned int row_b = 0; row_b < params.n; ++row_b) {
      for (unsigned int k = 0; k < params.k; k += kProcessorWordSize) {
        unsigned int upper_bound = min(k + kProcessorWordSize, params.k);

        uint64_t processor_word_a = get_aligned_k_bits(
            params.matrix_a, row_a * params.k, (row_a + 1) * params.k);
        uint64_t processor_word_b = get_aligned_k_bits(
            params.matrix_b, row_b * params.k, (row_b + 1) * params.k);

        if ((processor_word_a & processor_word_b) == 0) {
          continue;
        } else {
          set_bit((unsigned long long *)params.matrix_c,
                  (unsigned long long)row_a * params.n + row_b);
          break;
        }
      }
    }
  }
}

static __global__ void matrix_count_kernel(ParametersForMatrixBitCount params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int i = tid; i <= WORD_OFFSET(params.size); i += step) {
    unsigned long long x = params.data[i];
    x = (x & (0x5555555555555555)) + ((x >> 1) & (0x5555555555555555));
    x = (x & (0x3333333333333333)) + ((x >> 2) & (0x3333333333333333));
    x = (x & (0x0f0f0f0f0f0f0f0f)) + ((x >> 4) & (0x0f0f0f0f0f0f0f0f));
    x = (x & (0x00ff00ff00ff00ff)) + ((x >> 8) & (0x00ff00ff00ff00ff));
    x = (x & (0x0000ffff0000ffff)) + ((x >> 16) & (0x0000ffff0000ffff));
    x = (x & (0x00000000ffffffff)) + ((x >> 32) & (0x00000000ffffffff));
    atomicAdd(params.count, x);
  }
}

static __global__ void
init_bit_tiled_matrix_metadata(ParametersInitBitTiledMatrix params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int i = tid; i < params.tile_size; i += step) {
    for (unsigned int j = 0; j < params.tile_size; j++) {
      if (get_bit((uint64_t *)params.layout_matrix,
                  (uint64_t)(i * params.tile_size + j))) {
        unsigned long long pre_element_count_val =
            pre_element_count((uint64_t *)params.layout_matrix,
                              (uint64_t)i * params.tile_size + j);
        params.tile_col_idx[pre_element_count_val] = i;
        params.tile_row_idx[pre_element_count_val] = j;
        atomicAdd(params.tile_offset_row + i, (unsigned)1);
      }
    }
  }
}

void MatrixOperationsKernelWrapper::MatrixBitAnd(
    const hipStream_t &stream,
    const data_structures::DeviceOwnedBuffer<uint64_t> &matrix_a_buf,
    const data_structures::DeviceOwnedBuffer<uint64_t> &matrix_b_buf,
    data_structures::DeviceOwnedBuffer<uint64_t> *matrix_c_buf, uint32_t m,
    uint32_t k, uint32_t n) {
  dim3 dimBlock(64);
  dim3 dimGrid(64);

  ParametersForMatrixBitAnd params{.matrix_a = matrix_a_buf.GetPtr(),
                                   .matrix_b = matrix_b_buf.GetPtr(),
                                   .matrix_c = matrix_c_buf->GetPtr(),
                                   .m = m,
                                   .k = k,
                                   .n = n};

  matrix_and_kernel<<<dimBlock, dimGrid, 0, stream>>>(params);
}

void MatrixOperationsKernelWrapper::MatrixBitCount(
    const hipStream_t &stream,
    const data_structures::DeviceOwnedBuffer<uint64_t> &matrix_buf,
    data_structures::DeviceOwnedBuffer<uint64_t> *count_buf, uint64_t size) {

  dim3 dimBlock(64);
  dim3 dimGrid(64);
  ParametersForMatrixBitCount params{
      .data = reinterpret_cast<unsigned long long *>(matrix_buf.GetPtr()),
      .count = reinterpret_cast<unsigned long long *>(count_buf->GetPtr()),
      .size = size};
  matrix_count_kernel<<<dimBlock, dimGrid, 0, stream>>>(params);
}

void MatrixOperationsKernelWrapper::InitBitTiledMatrixMetadataByLayoutMatrix(
    const hipStream_t &stream,
    const data_structures::DeviceOwnedBuffer<uint64_t> &layout_matrix,
    data_structures::DeviceOwnedBuffer<uint32_t> *tile_offset_row,
    data_structures::DeviceOwnedBuffer<uint32_t> *tile_row_idx,
    data_structures::DeviceOwnedBuffer<uint32_t> *tile_col_idx,
    uint32_t tile_size) {

  std::cout << "InitBitTiledMatrixedByLayoutMatrix" << std::endl;
  dim3 dimBlock(1);
  dim3 dimGrid(1);
  ParametersInitBitTiledMatrix params{
      .layout_matrix =
          reinterpret_cast<unsigned long long *>(layout_matrix.GetPtr()),
      .tile_offset_row =
          reinterpret_cast<unsigned *>(tile_offset_row->GetPtr()),
      .tile_row_idx = reinterpret_cast<unsigned *>(tile_row_idx->GetPtr()),
      .tile_col_idx = reinterpret_cast<unsigned *>(tile_col_idx->GetPtr()),
      .tile_size = tile_size};

  init_bit_tiled_matrix_metadata<<<dimBlock, dimGrid, 0, stream>>>(params);
}

} // namespace kernel
} // namespace task
} // namespace core
} // namespace matrixgraph
} // namespace sics