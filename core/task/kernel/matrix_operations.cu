#include "hip/hip_runtime.h"
#include "core/task/kernel/matrix_operations.cuh"

#include <hip/hip_runtime.h>
#include <iostream>

#include "core/util/cuda_check.cuh"

namespace sics {
namespace matrixgraph {
namespace core {
namespace task {
namespace kernel {

#define WORD_OFFSET(i) (i >> 6)
#define BIT_OFFSET(i) (i & 0x3f)
#define min(a, b) ((a) < (b) ? (a) : (b))
#define max(a, b) ((a) > (b) ? (a) : (b))

static const uint32_t kProcessorWordSize = 64;

struct ParametersForMatrixBitAnd {
  uint64_t *matrix_a;
  uint64_t *matrix_b;
  uint64_t *matrix_c;
  uint64_t m;
  uint64_t k;
  uint64_t n;
};

struct ParametersForMatrixBitCount {
  unsigned long long *data;
  unsigned long long *count;
  unsigned long long size;
};

struct ParametersInitBitTiledMatrix {
  unsigned long long *layout_matrix;
  unsigned *tile_offset_row;
  unsigned *tile_row_idx;
  unsigned *tile_col_idx;
  unsigned long tile_size;
};

struct ParametersFillTiles {
  unsigned long tile_size;
  unsigned long n_strips;
  unsigned long n_nz_tile_a;
  unsigned long n_nz_tile_b;
  unsigned long n_nz_tile_c;
  unsigned long tile_unit;
  unsigned long tile_buffer_size;
  unsigned long long *layout_matrix_c;
  unsigned *tile_offset_row_a;
  unsigned *tile_offset_row_b;
  unsigned *tile_offset_row_c;
  unsigned *tile_row_idx_a;
  unsigned *tile_row_idx_b;
  unsigned *tile_row_idx_c;
  unsigned *tile_col_idx_a;
  unsigned *tile_col_idx_b;
  unsigned *tile_col_idx_c;
  unsigned long long *data_a;
  unsigned long long *data_b;
  unsigned long long *data_c;
};

__device__ static inline uint64_t get_bit(uint64_t *data, size_t i,
                                          size_t size) {
  if (i > size)
    return 0;
  return data[WORD_OFFSET(i)] & (1ull << BIT_OFFSET(i));
}

__device__ static inline bool drop() { return false; }

__device__ static inline uint64_t
get_aligned_k_bits(uint64_t *data, uint64_t start, uint64_t end) {
  uint64_t start_word = WORD_OFFSET(start);
  uint64_t end_word = WORD_OFFSET(end);

  uint64_t start_bit = BIT_OFFSET(start);
  uint64_t end_bit = BIT_OFFSET(end);

  uint64_t result = 0;
  uint64_t mask = ((1ull << (end_bit - start_bit)) - 1) << start_bit;

  result = (data[start_word] & mask) >> start_bit;
  return result;
}

__device__ static inline uint64_t set_bit(unsigned long long *data,
                                          unsigned long long i) {
  atomicOr(data + WORD_OFFSET(i), (1ull << BIT_OFFSET(i)));
}

__device__ static inline bool get_bit(uint64_t *data, uint64_t i) {
  return data[WORD_OFFSET(i)] & (1ull << BIT_OFFSET(i));
}

__device__ static inline size_t pre_element_count(const uint64_t *data,
                                                  uint64_t idx) {

  size_t count = 0;
  size_t bm_size = WORD_OFFSET(idx);
  size_t idx_offset = WORD_OFFSET(idx);
  size_t idx_bit_offset = BIT_OFFSET(idx);

  for (size_t i = 0; i <= bm_size; i++) {
    uint64_t x = 0;
    if (i == idx_offset) {
      uint64_t mask = (1ul << idx_bit_offset) - 1;
      x = data[i] & mask;
    } else {
      x = data[i];
    }
    x = (x & (0x5555555555555555)) + ((x >> 1) & (0x5555555555555555));
    x = (x & (0x3333333333333333)) + ((x >> 2) & (0x3333333333333333));
    x = (x & (0x0f0f0f0f0f0f0f0f)) + ((x >> 4) & (0x0f0f0f0f0f0f0f0f));
    x = (x & (0x00ff00ff00ff00ff)) + ((x >> 8) & (0x00ff00ff00ff00ff));
    x = (x & (0x0000ffff0000ffff)) + ((x >> 16) & (0x0000ffff0000ffff));
    x = (x & (0x00000000ffffffff)) + ((x >> 32) & (0x00000000ffffffff));
    count += (size_t)x;
  }

  return count;
}

__device__ static inline bool single_thread_matrix_bit_and(
    unsigned long tile_size, unsigned long long *matrix_a,
    unsigned long long *matrix_b, unsigned long long *matrix_c) {
  for (unsigned int row_a = 0; row_a < tile_size; ++row_a) {
    for (unsigned int row_b = 0; row_b < tile_size; ++row_b) {
      for (unsigned int k = 0; k < tile_size; k += kProcessorWordSize) {
        uint64_t processor_word_a = get_aligned_k_bits(
            (uint64_t *)matrix_a, row_a * tile_size, (row_a + 1) * tile_size);
        uint64_t processor_word_b = get_aligned_k_bits(
            (uint64_t *)matrix_b, row_b * tile_size, (row_b + 1) * tile_size);

        if ((processor_word_a & processor_word_b) == 0) {
          continue;
        } else {
          set_bit((unsigned long long *)matrix_c,
                  (unsigned long long)row_a * tile_size + row_b);
          break;
        }
      }
    }
  }
}

__device__ static inline void
find_intersection(unsigned size_l, unsigned size_r, unsigned *data_l,
                  unsigned *data_r, unsigned *data_out_l, unsigned *data_out_r,
                  unsigned *n_intersections) {
  unsigned i = 0;
  unsigned j = 0;

  while (i < size_l && j < size_r) {
    if (data_l[i] < data_r[j]) {
      i++;
    } else if (data_l[i] > data_r[j]) {
      j++;
    } else {
      data_out_l[*n_intersections] = i;
      data_out_r[*n_intersections] = j;
      (*n_intersections)++;
      i++;
      j++;
    }
  }
}

static __global__ void matrix_and_kernel(ParametersForMatrixBitAnd params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int row_a = tid; row_a < params.m; row_a += step) {
    for (unsigned int row_b = 0; row_b < params.n; ++row_b) {
      for (unsigned int k = 0; k < params.k; k += kProcessorWordSize) {
        unsigned int upper_bound = min(k + kProcessorWordSize, params.k);

        uint64_t processor_word_a = get_aligned_k_bits(
            params.matrix_a, row_a * params.k, (row_a + 1) * params.k);
        uint64_t processor_word_b = get_aligned_k_bits(
            params.matrix_b, row_b * params.k, (row_b + 1) * params.k);

        if ((processor_word_a & processor_word_b) == 0) {
          continue;
        } else {
          set_bit((unsigned long long *)params.matrix_c,
                  (unsigned long long)row_a * params.n + row_b);
          break;
        }
      }
    }
  }
}

static __global__ void matrix_count_kernel(ParametersForMatrixBitCount params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int i = tid; i <= WORD_OFFSET(params.size); i += step) {
    unsigned long long x = params.data[i];
    x = (x & (0x5555555555555555)) + ((x >> 1) & (0x5555555555555555));
    x = (x & (0x3333333333333333)) + ((x >> 2) & (0x3333333333333333));
    x = (x & (0x0f0f0f0f0f0f0f0f)) + ((x >> 4) & (0x0f0f0f0f0f0f0f0f));
    x = (x & (0x00ff00ff00ff00ff)) + ((x >> 8) & (0x00ff00ff00ff00ff));
    x = (x & (0x0000ffff0000ffff)) + ((x >> 16) & (0x0000ffff0000ffff));
    x = (x & (0x00000000ffffffff)) + ((x >> 32) & (0x00000000ffffffff));
    atomicAdd(params.count, x);
  }
}

static __global__ void
init_bit_tiled_matrix_metadata_kernel(ParametersInitBitTiledMatrix params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int i = tid; i < params.tile_size; i += step) {
    for (unsigned int j = 0; j < params.tile_size; j++) {
      if (get_bit((uint64_t *)params.layout_matrix,
                  (uint64_t)(i * params.tile_size + j))) {
        unsigned long long pre_element_count_val =
            pre_element_count((uint64_t *)params.layout_matrix,
                              (uint64_t)i * params.tile_size + j);
        params.tile_row_idx[pre_element_count_val] = i;
        params.tile_col_idx[pre_element_count_val] = j;
        atomicAdd(params.tile_offset_row + i, (unsigned)1);
      }
    }
  }
}

static __global__ void fill_tiles_kernel(ParametersFillTiles params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  for (unsigned int i = tid; i < params.n_nz_tile_c; i += step) {
    unsigned int x = params.tile_row_idx_c[i];
    unsigned int y = params.tile_col_idx_c[i];

    unsigned int nz_tile_line_x =
        params.tile_offset_row_a[x + 1] - params.tile_offset_row_a[x];

    unsigned int nz_tile_line_y =
        params.tile_offset_row_b[y + 1] - params.tile_offset_row_b[y];

    unsigned int *idx_intersection_l =
        new unsigned int[min(nz_tile_line_x, nz_tile_line_y)]();
    unsigned int *idx_intersection_r =
        new unsigned int[min(nz_tile_line_x, nz_tile_line_y)]();
    unsigned int n_intersections = 0;

    find_intersection(nz_tile_line_x, nz_tile_line_y,
                      params.tile_col_idx_a + params.tile_offset_row_a[x],
                      params.tile_col_idx_b + params.tile_offset_row_b[y],
                      idx_intersection_l, idx_intersection_r, &n_intersections);
    for (int l = 0; l < n_intersections; l++) {
      // perform bit and between (params.tile_offset_row_a[x] +
      // idx_intersection_l[l])-th  tile and (params.tile_offset_row_b[y] +
      // idx_intersection_r[r])-th tile.

      unsigned long long *matrix_a =
          params.data_a + params.tile_unit * (params.tile_offset_row_a[x] +
                                              idx_intersection_l[l]);
      unsigned long long *matrix_b =
          params.data_b + params.tile_unit * (params.tile_offset_row_b[y] +
                                              idx_intersection_r[l]);

      unsigned long long *matrix_c = params.data_c + params.tile_unit * i;

      single_thread_matrix_bit_and(params.tile_size, matrix_a, matrix_b,
                                   matrix_c);
    }

    delete[] idx_intersection_l;
    delete[] idx_intersection_r;
  }
}

void MatrixOperationsKernelWrapper::MatrixBitAnd(
    const hipStream_t &stream,
    const data_structures::DeviceOwnedBuffer<uint64_t> &matrix_a_buf,
    const data_structures::DeviceOwnedBuffer<uint64_t> &matrix_b_buf,
    data_structures::DeviceOwnedBuffer<uint64_t> *matrix_c_buf, uint32_t m,
    uint32_t k, uint32_t n) {
  dim3 dimBlock(64);
  dim3 dimGrid(64);

  ParametersForMatrixBitAnd params{.matrix_a = matrix_a_buf.GetPtr(),
                                   .matrix_b = matrix_b_buf.GetPtr(),
                                   .matrix_c = matrix_c_buf->GetPtr(),
                                   .m = m,
                                   .k = k,
                                   .n = n};

  matrix_and_kernel<<<dimBlock, dimGrid, 0, stream>>>(params);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    CUDA_CHECK(err);
  }
}

void MatrixOperationsKernelWrapper::MatrixBitCount(
    const hipStream_t &stream,
    const data_structures::DeviceOwnedBuffer<uint64_t> &matrix_buf,
    data_structures::DeviceOwnedBuffer<uint64_t> *count_buf, uint64_t size) {

  dim3 dimBlock(64);
  dim3 dimGrid(64);
  ParametersForMatrixBitCount params{
      .data = reinterpret_cast<unsigned long long *>(matrix_buf.GetPtr()),
      .count = reinterpret_cast<unsigned long long *>(count_buf->GetPtr()),
      .size = size};
  matrix_count_kernel<<<dimBlock, dimGrid, 0, stream>>>(params);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    CUDA_CHECK(err);
  }
}

void MatrixOperationsKernelWrapper::InitBitTiledMatrixMetadataByLayoutMatrix(
    const hipStream_t &stream,
    const data_structures::DeviceOwnedBuffer<uint64_t> &layout_matrix,
    data_structures::DeviceOwnedBuffer<uint32_t> *tile_offset_row,
    data_structures::DeviceOwnedBuffer<uint32_t> *tile_row_idx,
    data_structures::DeviceOwnedBuffer<uint32_t> *tile_col_idx,
    uint32_t tile_size) {

  dim3 dimBlock(64);
  dim3 dimGrid(64);
  ParametersInitBitTiledMatrix params{
      .layout_matrix =
          reinterpret_cast<unsigned long long *>(layout_matrix.GetPtr()),
      .tile_offset_row =
          reinterpret_cast<unsigned *>(tile_offset_row->GetPtr()),
      .tile_row_idx = reinterpret_cast<unsigned *>(tile_row_idx->GetPtr()),
      .tile_col_idx = reinterpret_cast<unsigned *>(tile_col_idx->GetPtr()),
      .tile_size = tile_size};

  init_bit_tiled_matrix_metadata_kernel<<<dimBlock, dimGrid, 0, stream>>>(
      params);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    CUDA_CHECK(err);
  }
}

void MatrixOperationsKernelWrapper::FillTiles(
    const hipStream_t &stream, size_t tile_size, size_t n_strips,
    size_t n_nz_tile_a, size_t n_nz_tile_b, size_t n_nz_tile_c,
    const data_structures::UnifiedOwnedBuffer<uint64_t> &layout_matrix_c,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_offset_row_a,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_offset_row_b,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_offset_row_c,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_row_idx_a,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_row_idx_b,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_row_idx_c,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_col_idx_a,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_col_idx_b,
    const data_structures::UnifiedOwnedBuffer<uint32_t> &tile_col_idx_c,
    const data_structures::UnifiedOwnedBuffer<uint64_t> &data_a,
    const data_structures::UnifiedOwnedBuffer<uint64_t> &data_b,
    data_structures::UnifiedOwnedBuffer<uint64_t> *data_c) {

  dim3 dimBlock(64);
  dim3 dimGrid(64);

  auto tile_unit = max(1u, (WORD_OFFSET(tile_size * tile_size)));
  auto tile_buffer_size =
      sizeof(uint64_t) * max(1u, WORD_OFFSET(tile_size * tile_size));

  ParametersFillTiles params{
      .tile_size = tile_size,
      .n_strips = n_strips,
      .n_nz_tile_a = n_nz_tile_a,
      .n_nz_tile_b = n_nz_tile_b,
      .n_nz_tile_c = n_nz_tile_c,
      .tile_unit = tile_unit,
      .tile_buffer_size = tile_buffer_size,
      .layout_matrix_c = (unsigned long long *)layout_matrix_c.GetPtr(),
      .tile_offset_row_a = tile_offset_row_a.GetPtr(),
      .tile_offset_row_b = tile_offset_row_b.GetPtr(),
      .tile_offset_row_c = tile_offset_row_c.GetPtr(),
      .tile_row_idx_a = tile_row_idx_a.GetPtr(),
      .tile_row_idx_b = tile_row_idx_b.GetPtr(),
      .tile_row_idx_c = tile_row_idx_c.GetPtr(),
      .tile_col_idx_a = tile_col_idx_a.GetPtr(),
      .tile_col_idx_b = tile_col_idx_b.GetPtr(),
      .tile_col_idx_c = tile_col_idx_c.GetPtr(),
      .data_a = (unsigned long long *)data_a.GetPtr(),
      .data_b = (unsigned long long *)data_b.GetPtr(),
      .data_c = (unsigned long long *)(data_c->GetPtr())};

  fill_tiles_kernel<<<dimBlock, dimGrid, 0, stream>>>(params);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    CUDA_CHECK(err);
  }
}

} // namespace kernel
} // namespace task
} // namespace core
} // namespace matrixgraph
} // namespace sics