#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

#include "core/common/consts.h"
#include "core/common/host_algorithms.cuh"
#include "core/common/types.h"
#include "core/data_structures/device_buffer.cuh"
#include "core/data_structures/host_buffer.cuh"
#include "core/data_structures/unified_buffer.cuh"
#include "core/task/kernel/data_structures/hash_buckets.cuh"
#include "core/task/kernel/data_structures/immutable_csr_gpu.cuh"
#include "core/task/kernel/data_structures/kernel_bitmap.cuh"
#include "core/task/kernel/data_structures/mini_kernel_bitmap.cuh"
#include "core/task/kernel/kernel_woj_subiso.cuh"

namespace sics {
namespace matrixgraph {
namespace core {
namespace task {
namespace kernel {

using EdgeIndex = sics::matrixgraph::core::common::EdgeIndex;
using VertexLabel = sics::matrixgraph::core::common::VertexLabel;
using VertexID = sics::matrixgraph::core::common::VertexID;
using sics::matrixgraph::core::common::kMaxNumCandidatesPerThread;
using sics::matrixgraph::core::common::kMaxNumWeft;
using sics::matrixgraph::core::common::kMaxVertexID;
using sics::matrixgraph::core::task::kernel::HostKernelBitmap;
using sics::matrixgraph::core::task::kernel::HostMiniKernelBitmap;
using sics::matrixgraph::core::task::kernel::KernelBitmap;
using sics::matrixgraph::core::task::kernel::MiniKernelBitmap;
using HashBuckets = sics::matrixgraph::core::task::kernel::HashBuckets;
using BufferUint8 = sics::matrixgraph::core::data_structures::Buffer<uint8_t>;
using BufferUint32 = sics::matrixgraph::core::data_structures::Buffer<uint32_t>;
using BufferVertexID =
    sics::matrixgraph::core::data_structures::Buffer<VertexID>;
using UnifiedOwnedBufferEdgeIndex =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<EdgeIndex>;
using UnifiedOwnedBufferVertexID =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<VertexID>;
using UnifiedOwnedBufferVertexLabel =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<VertexLabel>;
using UnifiedOwnedBufferUint8 =
    sics::matrixgraph::core::data_structures::UnifiedOwnedBuffer<uint8_t>;
using BufferVertexLabel =
    sics::matrixgraph::core::data_structures::Buffer<VertexLabel>;
using BufferVertexID =
    sics::matrixgraph::core::data_structures::Buffer<VertexID>;

struct LocalMatches {
  VertexID *data = nullptr;
  VertexID *size = nullptr;
};

struct ParametersFilter {
  VertexID u_eid;
  VertexID *exec_path_in_edges = nullptr;
  VertexID n_vertices_p;
  EdgeIndex n_edges_p;
  uint8_t *data_p;
  VertexLabel *v_label_p = nullptr;
  VertexID n_vertices_g;
  EdgeIndex n_edges_g;
  uint8_t *data_g = nullptr;
  VertexID *edgelist_g = nullptr;
  VertexLabel *v_label_g = nullptr;
  HashBuckets hash_buckets;
};

__forceinline__ __device__ unsigned lane_id() {
  unsigned ret;
  asm volatile("mov.u32 %0, %laneid;" : "=r"(ret));
  return ret;
}

__forceinline__ __device__ unsigned warp_id() {
  unsigned ret;
  asm volatile("mov.u32 %0, %warpid;" : "=r"(ret));
  return ret;
}

static __forceinline__ __device__ bool
LabelFilter(const ParametersFilter &params, VertexID u_idx, VertexID v_idx) {
  VertexID *globalid_g = (VertexID *)(params.data_g);
  VertexLabel v_label = params.v_label_g[globalid_g[v_idx]];
  VertexLabel u_label = params.v_label_p[u_idx];
  return u_label == v_label;
}

static __forceinline__ __device__ bool
LabelDegreeFilter(const ParametersFilter &params, VertexID u_idx,
                  VertexID v_idx) {

  VertexID *globalid_p = (VertexID *)(params.data_p);
  VertexID *in_degree_p = globalid_p + params.n_vertices_p;
  VertexID *out_degree_p = in_degree_p + params.n_vertices_p;

  VertexID *globalid_g = (VertexID *)(params.data_g);
  VertexID *in_degree_g = globalid_g + params.n_vertices_g;
  VertexID *out_degree_g = in_degree_g + params.n_vertices_g;

  VertexLabel v_label = params.v_label_g[globalid_g[v_idx]];
  VertexLabel u_label = params.v_label_p[u_idx];

  if (u_label != v_label) {
    return false;
  } else {
    return out_degree_g[v_idx] >= out_degree_p[u_idx];
  }
}

static __forceinline__ __device__ bool Filter(const ParametersFilter &params,
                                              VertexID u_idx, VertexID v_idx) {

  return LabelFilter(params, u_idx, v_idx);

  // return NeighborLabelCounterFilter(params, u_idx, v_idx);

  // return LabelDegreeFilter(params, u_idx, v_idx);
}

static __noinline__ __global__ void WOJExtendKernel(ParametersFilter params) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int step = blockDim.x * gridDim.x;

  enum : unsigned { warp_size = 32, log_warp_size = 5 };
  auto lane_id = threadIdx.x & (warp_size - 1);
  auto warp_id = threadIdx.x >> log_warp_size;

  __shared__ VertexID local_matches_data[2048];
  __shared__ VertexID local_matches_offset;

  VertexID u_eid = params.u_eid;
  VertexID u_src = params.exec_path_in_edges[2 * u_eid];
  VertexID u_dst = params.exec_path_in_edges[2 * u_eid + 1];
  for (VertexID e_idx = tid; e_idx < params.n_edges_g; e_idx += step) {
    VertexID v_src = params.edgelist_g[2 * e_idx];
    VertexID v_dst = params.edgelist_g[2 * e_idx + 1];
    bool src_tag = true;
    bool dst_tag = true;

    if (u_src != -1) {
      src_tag = Filter(params, u_src, v_src);
    }
    if (u_dst != -1) {
      dst_tag = Filter(params, u_dst, v_dst);
    }
    if (src_tag && dst_tag) {
      VertexID offset = atomicAdd(&local_matches_offset, 1);
      local_matches_data[2 * offset] = v_src;
      local_matches_data[2 * offset + 1] = v_dst;
    }
  }

  __syncthreads();
  if (threadIdx.x == 0) {
    auto offset =
        atomicAdd(&params.hash_buckets.offset_[u_eid], local_matches_offset);
    memcpy(params.hash_buckets.data_[u_eid] + 2 * offset, local_matches_data,
           sizeof(VertexID) * 2 * local_matches_offset);
  }
}

void WOJSubIsoKernelWrapper::Filter(const ImmutableCSR &p,
                                    const ImmutableCSR &g, const Edges &e,
                                    const ExecutionPlan &exec_plan) {
  std::cout << "Filter" << std::endl;
  auto parallelism = std::thread::hardware_concurrency();
  std::vector<size_t> worker(parallelism);
  std::mutex mtx;

  std::iota(worker.begin(), worker.end(), 0);
  auto step = worker.size();

  // Init Streams
  std::vector<hipStream_t> p_streams_vec;
  p_streams_vec.resize(p.get_num_outgoing_edges());
  std::for_each(std::execution::par, worker.begin(), worker.end(),
                [step, &p_streams_vec, &mtx](auto w) {
                  for (VertexID i = w; i < p_streams_vec.size(); i += step) {
                    //                 hipSetDevice(common::hash_function(i) %
                    //                 4);
                    hipStreamCreate(&p_streams_vec[i]);
                  }
                });

  // Init pattern.
  BufferUint8 data_p;
  BufferVertexLabel v_label_p;
  BufferVertexID buffer_exec_path;
  BufferVertexID buffer_exec_path_in_edges;

  data_p.data = p.GetGraphBuffer();
  data_p.size = sizeof(VertexID) * p.get_num_vertices() +
                sizeof(VertexID) * p.get_num_vertices() +
                sizeof(VertexID) * p.get_num_vertices() +
                sizeof(EdgeIndex) * (p.get_num_vertices() + 1) +
                sizeof(EdgeIndex) * (p.get_num_vertices() + 1) +
                sizeof(VertexID) * p.get_num_incoming_edges() +
                sizeof(VertexID) * p.get_num_outgoing_edges() +
                sizeof(VertexID) * (p.get_max_vid() + 1);

  buffer_exec_path.data = exec_plan.get_exec_path_ptr();
  buffer_exec_path.size = sizeof(VertexID) * p.get_num_vertices();

  buffer_exec_path_in_edges.data = exec_plan.get_exec_path_in_edges_ptr();
  buffer_exec_path_in_edges.size = sizeof(VertexID) * p.get_num_vertices() * 2;

  v_label_p.data = p.GetVLabelBasePointer();
  v_label_p.size = sizeof(VertexLabel) * p.get_num_vertices();

  // Init data_graph.
  BufferUint8 data_g;
  BufferVertexLabel v_label_g;
  BufferVertexID data_edgelist_g;

  data_g.data = g.GetGraphBuffer();
  data_g.size = sizeof(VertexID) * g.get_num_vertices() +
                sizeof(VertexID) * g.get_num_vertices() +
                sizeof(VertexID) * g.get_num_vertices() +
                sizeof(EdgeIndex) * (g.get_num_vertices() + 1) +
                sizeof(EdgeIndex) * (g.get_num_vertices() + 1) +
                sizeof(VertexID) * g.get_num_incoming_edges() +
                sizeof(VertexID) * g.get_num_outgoing_edges() +
                sizeof(VertexID) * (g.get_max_vid() + 1);

  v_label_g.data = g.GetVLabelBasePointer();
  v_label_g.size = sizeof(VertexLabel) * g.get_num_vertices();

  data_edgelist_g.data = (VertexID *)e.get_base_ptr();
  data_edgelist_g.size = sizeof(VertexID) * e.get_metadata().num_edges * 2;

  //  Init output.
  std::vector<HashBuckets> hash_buckets_vec;
  hash_buckets_vec.resize(4);

  std::vector<ImmutableCSRGPU> data_graph_gpu_vec;
  data_graph_gpu_vec.resize(4);
  std::vector<ImmutableCSRGPU> pattern_graph_gpu_vec;
  pattern_graph_gpu_vec.resize(4);

  std::vector<UnifiedOwnedBufferVertexID> exec_path_in_edges_vec;
  exec_path_in_edges_vec.resize(4);
  std::vector<UnifiedOwnedBufferUint8> data_p_vec;
  data_p_vec.resize(4);
  std::vector<UnifiedOwnedBufferVertexLabel> v_label_p_vec;
  v_label_p_vec.resize(4);
  std::vector<UnifiedOwnedBufferUint8> data_g_vec;
  data_g_vec.resize(4);
  std::vector<UnifiedOwnedBufferVertexID> edgelist_g_vec;
  edgelist_g_vec.resize(4);
  std::vector<UnifiedOwnedBufferVertexLabel> v_label_g_vec;
  v_label_g_vec.resize(4);

  for (VertexID _ = 0; _ < 4; _++) {
    data_graph_gpu_vec[_].Init(g);
    pattern_graph_gpu_vec[_].Init(p);
    exec_path_in_edges_vec[_].Init(buffer_exec_path_in_edges);
    data_p_vec[_].Init(data_p);
    v_label_p_vec[_].Init(v_label_p);
    data_g_vec[_].Init(data_g);
    edgelist_g_vec[_].Init(data_edgelist_g);
    v_label_g_vec[_].Init(v_label_g);
    hash_buckets_vec[_].Init(p.get_num_outgoing_edges(), kMaxNumWeft);
  }

  dim3 dimBlock(1);
  dim3 dimGrid(1);

  auto time1 = std::chrono::system_clock::now();
  for (VertexID _ = 0; _ < p.get_num_outgoing_edges(); _++) {
    VertexID device_id = common::hash_function(_) % 4;
    //  hipSetDevice(device_id);
    hipStream_t &stream = p_streams_vec[_];
    ParametersFilter params{
        .u_eid = _,
        .exec_path_in_edges = exec_path_in_edges_vec[device_id].GetPtr(),
        .n_vertices_p = p.get_num_vertices(),
        .n_edges_p = p.get_num_outgoing_edges(),
        .data_p = data_p_vec[device_id].GetPtr(),
        .v_label_p = v_label_p_vec[device_id].GetPtr(),
        .n_vertices_g = g.get_num_vertices(),
        .n_edges_g = g.get_num_outgoing_edges(),
        .data_g = data_g_vec[device_id].GetPtr(),
        .edgelist_g = edgelist_g_vec[device_id].GetPtr(),
        .v_label_g = v_label_g_vec[device_id].GetPtr(),
        .hash_buckets = hash_buckets_vec[0],
    };
    // for (auto __ = 0; __ < p.get_num_outgoing_edges(); __++) {
    //   std::cout << params.exec_path_in_edges[2 * __]
    //             << params.exec_path_in_edges[2 * __ + 1] << std::endl;
    // }
    WOJExtendKernel<<<dimGrid, dimBlock, 0, stream>>>(params);
  }

  hipDeviceSynchronize();

  auto time2 = std::chrono::system_clock::now();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    CUDA_CHECK(err);
  }

  for (auto j = 0; j < p.get_num_vertices(); j++) {
    std::cout << ", offset: " << hash_buckets_vec[0].offset_[j] << std::endl;
    for (VertexID eid = 0; eid < hash_buckets_vec[0].offset_[j]; eid++) {
      std::cout << hash_buckets_vec[0].data_[j][2 * eid] << " -> "
                << hash_buckets_vec[0].data_[j][2 * eid + 1] << std::endl;
    }
  }
  std::cout << "-------------" << std::endl;

  std::cout << "[Filter]:"
            << std::chrono::duration_cast<std::chrono::microseconds>(time2 -
                                                                     time1)
                       .count() /
                   (double)CLOCKS_PER_SEC
            << std::endl;

  for (VertexID _ = 0; _ < 4; _++) {
    pattern_graph_gpu_vec[_].Free();
    data_graph_gpu_vec[_].Free();
  }
}

} // namespace kernel
} // namespace task
} // namespace core
} // namespace matrixgraph
} // namespace sics